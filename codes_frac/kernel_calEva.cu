#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"
#include <iostream>
#include "cu_PWA_PARAS.h"
#include <vector>
#include <fstream>
#include <math.h>
#include "cu_DPFPropogator.h"
#include "kernel_calEva.h"
#include <assert.h>

using namespace std;

#define CUDA_CALL(x) {const hipError_t a=(x); if(a != hipSuccess) {printf("\nerror in line:%d CUDAError:%s(err_num=%d)\n",__LINE__,hipGetErrorString(a),a); hipDeviceReset(); assert(0); }}
 


 __device__ float calEva(const cu_PWA_PARAS *pp, const int * parameter , float2 * complex_para ,const float * d_paraList,float *d_mlk,int idp) 
    ////return square of complex amplitude
{
    //	static int A=0;
    //	A++;
    
    int _N_spinList     =parameter[0];
    int _N_massList     =parameter[1];
    int _N_mass2List    =parameter[2];
    int _N_widthList    =parameter[3];
    int _N_g1List       =parameter[4];
    int _N_g2List       =parameter[5];
    int _N_b1List       =parameter[6];
    int _N_b2List       =parameter[7];
    int _N_b3List       =parameter[8];
    int _N_b4List       =parameter[9];
    int _N_b5List       =parameter[10];
    int _N_rhoList      =parameter[11];
    int _N_fracList     =parameter[12];
    int _N_phiList      =parameter[13];
    int _N_propList     =parameter[14];
    const int const_nAmps=parameter[15];
    float value = 0.;
    //float2 fCF[const_nAmps][4];
    float2 *fCF=complex_para; 
    //float2 (*fCF)[4]=(float2 (*)[4])malloc(sizeof(float2)*const_nAmps*4);
    //float2 fCP[const_nAmps];
    //float2 * fCP=(float2 *)malloc(sizeof(float2)*const_nAmps);
    float2 * fCP=&complex_para[4*const_nAmps];
    float2 * crp1=&complex_para[5*const_nAmps];
    float2 * crp11=&complex_para[6*const_nAmps];


    //float2 pa[const_nAmps][const_nAmps];
    float2 * pa=&complex_para[7*const_nAmps];
    float2 * fu=&complex_para[(7+const_nAmps)*const_nAmps];


    /*float2 **pa,**fu;
    pa=(float2 **)malloc(sizeof(float2 *)*const_nAmps);
    fu=(float2 **)malloc(sizeof(float2 *)*const_nAmps);
    for(int i=0;i<const_nAmps;i++)
    {
        pa[i]=(float2 *)malloc(sizeof(float2)*const_nAmps);
        fu[i]=(float2 *)malloc(sizeof(float2)*const_nAmps);
    }
    //float2 fu[const_nAmps][const_nAmps];
    //float2 crp1[const_nAmps];
    float2 * crp1=(float2 *)malloc(sizeof(float2)*const_nAmps);
    //float2 crp11[const_nAmps];
    float2 * crp11=(float2 *)malloc(sizeof(float2)*const_nAmps);
    */
    float2 cr0p11;
    //float2 ca2p1;
    float2 cw2p11;
    float2 cw2p12;
    float2 cw2p15;
    float2 cw;
    float2 c1p12_12,c1p13_12,c1p12_13,c1p13_13,c1p12_14,c1p13_14;
    float2 cr1m12_1,cr1m13_1;
    float2 crpf1,crpf2;

    for(int index=0; index<const_nAmps; index++) {
        float rho0 = d_paraList[_N_rhoList++];
        float frac0 = d_paraList[_N_fracList++];
        float phi0 = d_paraList[_N_phiList++];
        int spin_now = d_paraList[_N_spinList++];
        int propType_now = d_paraList[_N_propList++];
    //cout<<"haha: "<< __LINE__ << endl;

        rho0 *= std::exp(frac0);
        fCP[index]=make_hipFloatComplex(rho0*std::cos(phi0),rho0*std::sin(phi0));
        //        //cout<<"fCP[index]="<<fCP[index]<<endl;
        //std::cout << __FILE__ << __LINE__ << " : " << propType_now << std::endl;
        switch(propType_now)
        {
            //  //cout<<"haha: "<< __LINE__ << endl;
            //                     ordinary  Propagator  Contribution
            case 1:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    float mass0 = d_paraList[_N_massList++];
                    float width0 = d_paraList[_N_widthList++];
                    //					//cout<<"mass0="<<mass0<<endl;
                    //					//cout<<"width0="<<width0<<endl;
                    crp1[index]=propogator(mass0,width0,pp->s23);
                }
                break;
            //	Flatte   Propagator Contribution
            case 2:
                {
                    //RooRealVar *g1 = (RooRealVar*)_g1IterV[omp_id]->Next();
                    //RooRealVar *g2 = (RooRealVar*)_g2IterV[omp_id]->Next();
                    float mass980 = d_paraList[_N_massList++];
                    float g10 = d_paraList[_N_g1List++];
                    float g20 = d_paraList[_N_g2List++];
                    //float g10=g1->getVal();
                    //float g20=g2->getVal();
     //               			//cout<<"mass980="<<mass980<<endl;
     //               			//cout<<"g10="<<g10<<endl;
     //               			//cout<<"g20="<<g20<<endl;
     //                           //cout<<"pp.s23="<<pp.s23<< endl;
                    crp1[index]=propogator980(mass980,g10,g20,pp->s23);
     //               			//cout<<"crp1[index]="<<crp1[index]<<endl;
                }
                break;
                // sigma  Propagator Contribution
            case 3:
                {
                    //RooRealVar *b1 = (RooRealVar*)_b1IterV[omp_id]->Next();
                    //RooRealVar *b2 = (RooRealVar*)_b2IterV[omp_id]->Next();
                    //RooRealVar *b3 = (RooRealVar*)_b3IterV[omp_id]->Next();
                    //RooRealVar *b4 = (RooRealVar*)_b4IterV[omp_id]->Next();
                    //RooRealVar *b5 = (RooRealVar*)_b5IterV[omp_id]->Next();
                    //float mass600=mass->getVal();
                    //float b10=b1->getVal();
                    //float b20=b2->getVal();
                    //float b30=b3->getVal();
                    //float b40=b4->getVal();
                    //float b50=b5->getVal();
                    float mass600 = d_paraList[_N_massList++];
                    float b10 = d_paraList[_N_b1List++];
                    float b20 = d_paraList[_N_b2List++];
                    float b30 = d_paraList[_N_b3List++];
                    float b40 = d_paraList[_N_b4List++];
                    float b50 = d_paraList[_N_b5List++];
                    crp1[index]=propogator600(mass600,b10,b20,b30,b40,b50,pp->s23);
                    //			//cout<<"crp1[index]3="<<crp1[index]<<endl;
                }
                break;
                // 1- or 1+  Contribution
            case 4:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //float mass0=mass->getVal();
                    //float width0=width->getVal();
                    float mass0 = d_paraList[_N_massList++];
                    float width0 = d_paraList[_N_widthList++];
                    crp1[index]=propogator(mass0,width0,pp->sv2);
                    crp11[index]=propogator(mass0,width0,pp->sv3);
                }
                break;
                //  phi(1650) f0(980) include flatte and ordinary Propagator joint Contribution
            case 5:
                {
                    //RooRealVar *mass2  = (RooRealVar*)_mass2IterV[omp_id]->Next();
                    //RooRealVar *g1 = (RooRealVar*)_g1IterV[omp_id]->Next();
                    //RooRealVar *g2 = (RooRealVar*)_g2IterV[omp_id]->Next();
                    //float mass980=mass2->getVal();
                    //float g10=g1->getVal();
                    //float g20=g2->getVal();
                    float mass980 = d_paraList[_N_mass2List++];
                    float g10 = d_paraList[_N_g1List++];
                    float g20 = d_paraList[_N_g2List++];
                    //					//cout<<"mass980="<<mass980<<endl;
                    //					//cout<<"g10="<<g10<<endl;
                    //					//cout<<"g20="<<g20<<endl;
                    crp1[index]=propogator980(mass980,g10,g20,pp->sv);
                    //					//cout<<"crp1[index]="<<crp1[index]<<endl;
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //float mass1680=mass->getVal();
                    //float width1680=width->getVal();
                    float mass1680 = d_paraList[_N_massList++];
                    float width1680 = d_paraList[_N_widthList++];
                    //					//cout<<"mass1680="<<mass1680<<endl;
                    //					//cout<<"width1680="<<width1680<<endl;
                    crp11[index]=propogator(mass1680,width1680,pp->s23);
                    //					//cout<<"crp11[index]="<<crp11[index]<<endl;
                }
                break;
            case 6:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //float mass0=mass->getVal();
                    //float width0=width->getVal();
                    float mass0 = d_paraList[_N_massList++];
                    float width0 = d_paraList[_N_widthList++];
                    //					//cout<<"mass0="<<mass0<<endl;
                    //					//cout<<"width0="<<width0<<endl;
                    crp1[index]=propogator1270(mass0,width0,pp->s23);
                    //			//cout<<"crp1[index]6="<<crp1[index]<<endl;
                }
            default :
                ;
        }
    //cout << "LINE: " << __LINE__ << endl;
        for(int i=0;i<2;i++){
            ////cout<<"haha: "<< __LINE__ << endl;
            //		//cout<<"spin_now="<<spin_now<<endl;
            switch(spin_now)
            {
                case 11:
                    //1+_1 contribution
                    //fCF[index][i]=pp.w1p12_1[i]*crp1[index]+pp.w1p13_1[i]*crp11[i];
                    fCF[index*4+i]=hipCaddf( cuCmulfc(pp->w1p12_1[i],crp1[index]),cuCmulfc(pp->w1p13_1[i],crp11[i]) );

                    break;
                case 12:
                    //1+_2 contribution
                    //c1p12_12=crp1[index]/pp.b2qbv2;
                    c1p12_12=cuCdivcf(crp1[index],pp->b2qbv2);
                    //c1p13_12=crp11[index]/pp.b2qbv3;
                    c1p13_12=cuCdivcf(crp11[index],pp->b2qbv3);
                    //fCF[index][i]=pp.w1p12_2[i]*c1p12_12+pp.w1p13_2[i]*c1p13_12;
                    fCF[index*4+i]=hipCaddf( cuCmulfc(pp->w1p12_2[i],c1p12_12) , cuCmulfc(pp->w1p13_2[i],c1p13_12) );
                
                    break;
                case 13:
                    //1+_3 contribution
                    //c1p12_13=crp1[index]/pp.b2qjv2;
                    c1p12_13=cuCdivcf(crp1[index],pp->b2qjv2);
                    //c1p13_13=crp11[index]/pp.b2qjv3;
                    c1p13_13=cuCdivcf(crp11[index],pp->b2qjv3);
                    //fCF[index][i]=pp.w1p12_3[i]*c1p12_13+pp.w1p13_3[i]*c1p13_13;
                    fCF[index*4+i]=hipCaddf( cuCmulfc(pp->w1p12_3[i],c1p12_13) , cuCmulfc(pp->w1p13_3[i],c1p13_13) );

                    break;
                case 14:
                    //1+_4 contribution
                    //c1p12_12=crp1[index]/pp.b2qbv2;
                    c1p12_12=cuCdivcf(crp1[index],pp->b2qbv2);
                    
                    c1p13_12=cuCdivcf(crp11[index],pp->b2qbv3);
                    c1p12_14=cuCdivcf(c1p12_12,pp->b2qjv2);
                    c1p13_14=cuCdivcf(c1p13_12,pp->b2qjv3);
                    fCF[index*4+i]=hipCaddf( cuCmulfc(pp->w1p12_4[i],c1p12_14), cuCmulfc(pp->w1p13_4[i],c1p13_14));

                    break;
                case 111:
                    //1-__1 contribution
                    cr1m12_1=cuCdivcf( cuCdivcf(crp1[index],pp->b1qjv2) , pp->b1qbv2);
                    cr1m13_1=cuCdivcf( cuCdivcf(crp11[index],pp->b1qjv3) , pp->b1qbv3);
                    fCF[index*4+i]=hipCaddf( cuCmulfc(pp->w1m12[i],cr1m12_1), cuCmulfc(pp->w1m13[i],cr1m13_1));

                    break;
                case 191:
                    //phi(1650)f0(980)_1 contribution
                    //		//cout<<"b1q2r23="<<b1q2r23<<endl;
                    crpf1=cuCdivcf( hipCmulf(crp1[index],crp11[index]),pp->b1q2r23 );
                    //		//cout<<"crpf1="<<crpf1<<endl;
                    fCF[index*4+i]=cuCmulfc(pp->ak23w[i],crpf1);
                    //	//cout<<"fCF[index][i]="<<fCF[index][i]<<endl;

                    break;
                case 192:
                    //phi(1650)f0(980)_2 contribution
                    crpf1=cuCdivcf( hipCmulf(crp1[index],crp11[index]) , pp->b1q2r23);
                    crpf2=cuCdivcf(crpf1,pp->b2qjvf2);
                    fCF[index*4+i]=cuCmulfc(pp->wpf22[i],crpf2);

                    break;
                case 1:
                    //  //cout<<"haha: "<< __LINE__ << endl;
                    //01 contribution
                    //	//cout<<"wu[i]="<<wu[i]<<endl;
                    //	//cout<<"crp1[index]="<<crp1[index]<<endl;
                    //	//cout<<"index="<<index<<endl;
                    fCF[index*4+i]=cuCmulfc(pp->wu[i],crp1[index]);
                    //	//cout<<"fCF[index][i]="<<fCF[index][i]<<endl;
                    //	//cout<<"i="<<i<<endl;

                    break;
                case 2:
                    //02 contribution
                    cr0p11=cuCdivcf(crp1[index],pp->b2qjvf2);
                    fCF[index*4+i]=cuCmulfc(pp->w0p22[i],cr0p11);
                    //	//cout<<"fCF[index][i]02="<<fCF[index][i]<<endl;

                    break;
                case 21:
                    //21 contribution
                    //	//cout<<"b2qf2xx="<<b2qf2xx<<endl;
                    cw2p11=cuCdivcf(crp1[index],pp->b2qf2xx);
                    //	//cout<<"cw2p11="<<cw2p11<<endl;
                    //	//cout<<"w2p1[0]="<<w2p1[0]<<endl;
                    //	//cout<<"w2p1[1]="<<w2p1[1]<<endl;
                    fCF[index*4+i]=cuCmulfc(pp->w2p1[i],cw2p11);
                    //	//cout<<"fCF[index][i]21="<<fCF[index][i]<<endl;

                    break;
                case 22:
                    //22 contribution
                    cw2p11=cuCdivcf(crp1[index],pp->b2qf2xx);
                    cw2p12=cuCdivcf(cw2p11,pp->b2qjvf2);
                    fCF[index*4+i]=cuCmulfc(pp->w2p2[i],cw2p12);

                    break;
                case 23:
                    //23 contribution
                    cw2p11=cuCdivcf(crp1[index],pp->b2qf2xx);
                    cw2p12=cuCdivcf(cw2p11,pp->b2qjvf2);
                    fCF[index*4+i]=cuCmulfc(pp->w2p3[i],cw2p12);

                    break;
                case 24:
                    //24 contribution
                    cw2p11=cuCdivcf(crp1[index],pp->b2qf2xx);
                    cw2p12=cuCdivcf(cw2p11,pp->b2qjvf2);
                    fCF[index*4+i]=cuCmulfc(pp->w2p4[i],cw2p12);

                    break;
                case 25:
                    //25 contribution
                    cw2p11=cuCdivcf(crp1[index],pp->b2qf2xx);
                    cw2p15=cuCdivcf(cw2p11,pp->b4qjvf2);
                    fCF[index*4+i]=cuCmulfc(pp->w2p5[i],cw2p15);

                default:		;
            }
        }

    }
    float carry(0);
    //#pragmaint  omp parallel for reduction(+:value)
    for(int i=0;i<const_nAmps;i++){
        //  //cout<<"haha: "<< __LINE__ << endl;    int mlk_cro_size=sizeof(float)*numElements
        for(int j=0;j<const_nAmps;j++){
            cw=hipCmulf(fCP[i],hipConjf(fCP[j]));
            //    //cout<<"cw="<<cw<<endl;
            if(i==j) pa[i*const_nAmps+j]=make_hipFloatComplex(hipCrealf(cw),0.0);
            else if(i<j) pa[i*const_nAmps+j]=make_hipFloatComplex(2*hipCrealf(cw),0.0);
            else pa[i*const_nAmps+j]=make_hipFloatComplex(0.0,2*hipCimagf(cw));
            cw=make_hipFloatComplex(0.0,0.0);
            for(int k=0;k<2;k++){
                cw=hipCaddf(cw,cuCdivcf( hipCmulf( fCF[i*4+k],hipConjf(fCF[j*4+k]) ),(float)2.0) );
                //   //cout<<"cwfu="<<cw<<endl;

            }
            if(i<=j) fu[i*const_nAmps+j]=make_hipFloatComplex(hipCrealf(cw),0.0);
            if(i>j) fu[i*const_nAmps+j]=make_hipFloatComplex(0.0,-hipCimagf(cw));
            //      //cout<<"pa[i][j]="<<pa[i][j]<<endl;
            //      //cout<<"fu[i][j]="<<fu[i][j]<<endl;
            float temp = hipCrealf( hipCmulf(pa[i*const_nAmps+j],fu[i*const_nAmps+j]) );//i have a big change here 
            float y = temp - carry;
            float t = value + y;
            carry = (t - value) - y;

            value = t; // Kahan Summation
        }
    }

    for(int i=0;i<const_nAmps;i++){
        float2 cw=hipCmulf(fCP[i],hipConjf(fCP[i]));
        float pa=hipCrealf(cw);

        cw=make_hipFloatComplex(0.0,0.0);
        for(int k=0;k<2;k++){
            //cw+=fCF[i][k]*hipConjf(fCF[i][k])/(float)2.0;
            cw=hipCaddf(cw,cuCdivcf( hipCmulf( fCF[i*4+k],hipConjf(fCF[i*4+k]) ),(float)2.0) );
        }
        float fu=hipCrealf(cw);
        d_mlk[idp*const_nAmps+i] = pa * fu;
    }
    /*
    free(fCP);
    for(int i=0;i<const_nAmps;i++)
    {
        free(pa[i]);
        free(fu[i]);
        //free(fCF[i]);
    } 
    free(fCF);
    free(pa);
    free(fu);
    free(crp1);
    free(crp11);
*/
    return (value <= 0) ? 1e-20 : value;
}

__global__ void kernel_store_fx(const float * float_pp,const int *parameter,float2 * d_complex_para ,const float *d_paraList,float * d_fx,float *d_mlk,int numElements,int begin)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<numElements && i>= begin)
    {
        int pwa_paras_size = sizeof(cu_PWA_PARAS) / sizeof(float);
        cu_PWA_PARAS *pp = (cu_PWA_PARAS*)&float_pp[i*pwa_paras_size];
        float2 *complex_para=&d_complex_para[i*(2*parameter[15]+7)*parameter[15]];
        d_fx[i]=calEva(pp,parameter,complex_para,d_paraList,d_mlk,i);
        //printf("%dgpu :: %.7f\n",i,pp->wu[0]);
        //printf("\nfx[%d]:%f\n",i,d_fx[i]);
        //fx[i]=calEva(pp,parameter,d_paraList,i);
    }
    
    //if(i==1)
    //{
        //printf("pp[0]:%f pp[end]:%f parameter[0]:%d parameter[16]:%d paraList[0]:%f \n",float_pp[0],float_pp[numElements*sizeof(cu_PWA_PARAS)/sizeof(float)-1],parameter[0],parameter[16],d_paraList[0]);
    //}
}

int host_store_fx(float *d_float_pp,int *h_parameter,float *h_paraList,int para_size, float *h_fx,float * h_mlk,int numElements,int begin)
{
    float *d_fx;
    CUDA_CALL(hipMalloc((void **)&(d_fx),numElements * sizeof(float)));
    //std::cout << __LINE__ << endl;
    int *d_parameter;
    CUDA_CALL(hipMalloc((void **)&(d_parameter),18 * sizeof(int)));
    CUDA_CALL(hipMemcpy(d_parameter , h_parameter, 18*sizeof(int), hipMemcpyHostToDevice));
    //cout << "\nd_parameter[16]" <<h_parameter[16] << endl;
    //std::cout << __LINE__ << endl;
    //std::cout << "d_paralist[0]: "<< h_paraList[0] << std::endl;
    //std::cout << "paralist[0]: "<< paraList[0] << std::endl;
    float *d_paraList;
    CUDA_CALL(hipMalloc((void **)&(d_paraList),para_size * sizeof(float)));
    CUDA_CALL(hipMemcpy(d_paraList , h_paraList, para_size * sizeof(float), hipMemcpyHostToDevice));
    //cout << "\nd_paraList : " <<h_paraList[0] << endl;
    //std::cout << __LINE__ << endl;
    //init d_complex_para
    float2 * d_complex_para;
    CUDA_CALL(hipMalloc( (void**)&d_complex_para,(h_parameter[15]*2+7)*h_parameter[15]*numElements*sizeof(float2) ));
    //init mlk
    float *d_mlk=NULL;
    CUDA_CALL(hipMalloc( (void **)&(d_mlk),(h_parameter[16]+h_parameter[17])*h_parameter[15]*sizeof(float) ));
    //ut << "nAmps="<< h_parameter[15] << "iEnd=" << (h_parameter[16]+h_parameter[17]) << endl;
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    //printf("%d\n",sizeof(float2)*h_parameter[15]*(7+h_parameter[15])*numElements );
    kernel_store_fx<<<blocksPerGrid, threadsPerBlock>>>(d_float_pp, d_parameter,d_complex_para,d_paraList,d_fx,d_mlk, numElements,begin);
     //std::cout << __LINE__ << endl;
    CUDA_CALL(hipGetLastError());
    //CUDA_CALL(hipMemcpy(h_fx , d_fx, numElements * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_mlk , d_mlk, (h_parameter[16]+h_parameter[17])*h_parameter[15]*sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_fx , d_fx, numElements * sizeof(float), hipMemcpyDeviceToHost));

    //free memory
    //CUDA_CALL(hipFree(d_float_pp));
    CUDA_CALL(hipFree(d_fx));
    CUDA_CALL(hipFree(d_complex_para));
    CUDA_CALL(hipFree(d_parameter));
    CUDA_CALL(hipFree(d_paraList));
    CUDA_CALL(hipFree(d_mlk));

    //ofstream cout("data_fx_cal");
    //std::cout << __LINE__ << endl;
    //for(int i=begin;i<numElements;i++)
    //{
        //cout << h_fx[i] << endl;
    //}
    //cout.close();
    return 0;
}

void cu_malloc_h_pp(float *h_float_pp,float *&d_float_pp,int length)
{
    int array_size = sizeof(cu_PWA_PARAS) / sizeof(float) * length;
    int mem_size = array_size * sizeof(float);
    CUDA_CALL(hipMalloc((void **)&d_float_pp, mem_size));
    CUDA_CALL(hipMemcpy(d_float_pp , h_float_pp, mem_size, hipMemcpyHostToDevice));
}


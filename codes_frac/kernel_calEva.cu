#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"
#include <iostream>
#include "cu_PWA_PARAS.h"
#include <vector>
#include <fstream>
#include <math.h>
#include "cu_DPFPropogator.h"
#include "kernel_calEva.h"
#include <assert.h>

using namespace std;

#define CUDA_CALL(x) {const hipError_t a=(x); if(a != hipSuccess) {printf("\nerror in line:%d CUDAError:%s(err_num=%d)\n",__LINE__,hipGetErrorString(a),a); hipDeviceReset(); assert(0); }}

    


 __device__ double calEva(const cu_PWA_PARAS *pp, const int * parameter , const double * d_paraList,double *d_mlk,int idp) 
    ////return square of complex amplitude
{
    //	static int A=0;
    //	A++;
    
    int _N_spinList     =parameter[0];
    int _N_massList     =parameter[1];
    int _N_mass2List    =parameter[2];
    int _N_widthList    =parameter[3];
    int _N_g1List       =parameter[4];
    int _N_g2List       =parameter[5];
    int _N_b1List       =parameter[6];
    int _N_b2List       =parameter[7];
    int _N_b3List       =parameter[8];
    int _N_b4List       =parameter[9];
    int _N_b5List       =parameter[10];
    int _N_rhoList      =parameter[11];
    int _N_fracList     =parameter[12];
    int _N_phiList      =parameter[13];
    int _N_propList     =parameter[14];
    const int const_nAmps=parameter[15];
    double value = 0.;
    //double2 fCF[const_nAmps][4];
    double2 (*fCF)[4]=(double2 (*)[4])malloc(sizeof(double2)*const_nAmps*4);
    //double2 fCP[const_nAmps];
    double2 * fCP=(double2 *)malloc(sizeof(double2)*const_nAmps);
    //double2 pa[const_nAmps][const_nAmps];
    double2 **pa,**fu;
    pa=(double2 **)malloc(sizeof(double2 *)*const_nAmps);
    fu=(double2 **)malloc(sizeof(double2 *)*const_nAmps);
    for(int i=0;i<const_nAmps;i++)
    {
        pa[i]=(double2 *)malloc(sizeof(double2)*const_nAmps);
        fu[i]=(double2 *)malloc(sizeof(double2)*const_nAmps);
    }
    //double2 fu[const_nAmps][const_nAmps];
    //double2 crp1[const_nAmps];
    double2 * crp1=(double2 *)malloc(sizeof(double2)*const_nAmps);
    //double2 crp11[const_nAmps];
    double2 * crp11=(double2 *)malloc(sizeof(double2)*const_nAmps);
    double2 cr0p11;
    //double2 ca2p1;
    double2 cw2p11;
    double2 cw2p12;
    double2 cw2p15;
    double2 cw;
    double2 c1p12_12,c1p13_12,c1p12_13,c1p13_13,c1p12_14,c1p13_14;
    double2 cr1m12_1,cr1m13_1;
    double2 crpf1,crpf2;

    for(int index=0; index<const_nAmps; index++) {
        double rho0 = d_paraList[_N_rhoList++];
        double frac0 = d_paraList[_N_fracList++];
        double phi0 = d_paraList[_N_phiList++];
        int spin_now = d_paraList[_N_spinList++];
        int propType_now = d_paraList[_N_propList++];
    //cout<<"haha: "<< __LINE__ << endl;

        rho0 *= std::exp(frac0);
        fCP[index]=make_hipDoubleComplex(rho0*std::cos(phi0),rho0*std::sin(phi0));
        //        //cout<<"fCP[index]="<<fCP[index]<<endl;
        //std::cout << __FILE__ << __LINE__ << " : " << propType_now << std::endl;
        switch(propType_now)
        {
            //  //cout<<"haha: "<< __LINE__ << endl;
            //                     ordinary  Propagator  Contribution
            case 1:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    double mass0 = d_paraList[_N_massList++];
                    double width0 = d_paraList[_N_widthList++];
                    //					//cout<<"mass0="<<mass0<<endl;
                    //					//cout<<"width0="<<width0<<endl;
                    crp1[index]=propogator(mass0,width0,pp->s23);
                }
                break;
            //	Flatte   Propagator Contribution
            case 2:
                {
                    //RooRealVar *g1 = (RooRealVar*)_g1IterV[omp_id]->Next();
                    //RooRealVar *g2 = (RooRealVar*)_g2IterV[omp_id]->Next();
                    double mass980 = d_paraList[_N_massList++];
                    double g10 = d_paraList[_N_g1List++];
                    double g20 = d_paraList[_N_g2List++];
                    //double g10=g1->getVal();
                    //double g20=g2->getVal();
     //               			//cout<<"mass980="<<mass980<<endl;
     //               			//cout<<"g10="<<g10<<endl;
     //               			//cout<<"g20="<<g20<<endl;
     //                           //cout<<"pp.s23="<<pp.s23<< endl;
                    crp1[index]=propogator980(mass980,g10,g20,pp->s23);
     //               			//cout<<"crp1[index]="<<crp1[index]<<endl;
                }
                break;
                // sigma  Propagator Contribution
            case 3:
                {
                    //RooRealVar *b1 = (RooRealVar*)_b1IterV[omp_id]->Next();
                    //RooRealVar *b2 = (RooRealVar*)_b2IterV[omp_id]->Next();
                    //RooRealVar *b3 = (RooRealVar*)_b3IterV[omp_id]->Next();
                    //RooRealVar *b4 = (RooRealVar*)_b4IterV[omp_id]->Next();
                    //RooRealVar *b5 = (RooRealVar*)_b5IterV[omp_id]->Next();
                    //double mass600=mass->getVal();
                    //double b10=b1->getVal();
                    //double b20=b2->getVal();
                    //double b30=b3->getVal();
                    //double b40=b4->getVal();
                    //double b50=b5->getVal();
                    double mass600 = d_paraList[_N_massList++];
                    double b10 = d_paraList[_N_b1List++];
                    double b20 = d_paraList[_N_b2List++];
                    double b30 = d_paraList[_N_b3List++];
                    double b40 = d_paraList[_N_b4List++];
                    double b50 = d_paraList[_N_b5List++];
                    crp1[index]=propogator600(mass600,b10,b20,b30,b40,b50,pp->s23);
                    //			//cout<<"crp1[index]3="<<crp1[index]<<endl;
                }
                break;
                // 1- or 1+  Contribution
            case 4:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //double mass0=mass->getVal();
                    //double width0=width->getVal();
                    double mass0 = d_paraList[_N_massList++];
                    double width0 = d_paraList[_N_widthList++];
                    crp1[index]=propogator(mass0,width0,pp->sv2);
                    crp11[index]=propogator(mass0,width0,pp->sv3);
                }
                break;
                //  phi(1650) f0(980) include flatte and ordinary Propagator joint Contribution
            case 5:
                {
                    //RooRealVar *mass2  = (RooRealVar*)_mass2IterV[omp_id]->Next();
                    //RooRealVar *g1 = (RooRealVar*)_g1IterV[omp_id]->Next();
                    //RooRealVar *g2 = (RooRealVar*)_g2IterV[omp_id]->Next();
                    //double mass980=mass2->getVal();
                    //double g10=g1->getVal();
                    //double g20=g2->getVal();
                    double mass980 = d_paraList[_N_mass2List++];
                    double g10 = d_paraList[_N_g1List++];
                    double g20 = d_paraList[_N_g2List++];
                    //					//cout<<"mass980="<<mass980<<endl;
                    //					//cout<<"g10="<<g10<<endl;
                    //					//cout<<"g20="<<g20<<endl;
                    crp1[index]=propogator980(mass980,g10,g20,pp->sv);
                    //					//cout<<"crp1[index]="<<crp1[index]<<endl;
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //double mass1680=mass->getVal();
                    //double width1680=width->getVal();
                    double mass1680 = d_paraList[_N_massList++];
                    double width1680 = d_paraList[_N_widthList++];
                    //					//cout<<"mass1680="<<mass1680<<endl;
                    //					//cout<<"width1680="<<width1680<<endl;
                    crp11[index]=propogator(mass1680,width1680,pp->s23);
                    //					//cout<<"crp11[index]="<<crp11[index]<<endl;
                }
                break;
            case 6:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //double mass0=mass->getVal();
                    //double width0=width->getVal();
                    double mass0 = d_paraList[_N_massList++];
                    double width0 = d_paraList[_N_widthList++];
                    //					//cout<<"mass0="<<mass0<<endl;
                    //					//cout<<"width0="<<width0<<endl;
                    crp1[index]=propogator1270(mass0,width0,pp->s23);
                    //			//cout<<"crp1[index]6="<<crp1[index]<<endl;
                }
            default :
                ;
        }
    //cout << "LINE: " << __LINE__ << endl;
        for(int i=0;i<2;i++){
            ////cout<<"haha: "<< __LINE__ << endl;
            //		//cout<<"spin_now="<<spin_now<<endl;
            switch(spin_now)
            {
                case 11:
                    //1+_1 contribution
                    //fCF[index][i]=pp.w1p12_1[i]*crp1[index]+pp.w1p13_1[i]*crp11[i];
                    fCF[index][i]=hipCadd( cuCmuldc(pp->w1p12_1[i],crp1[index]),cuCmuldc(pp->w1p13_1[i],crp11[i]) );

                    break;
                case 12:
                    //1+_2 contribution
                    //c1p12_12=crp1[index]/pp.b2qbv2;
                    c1p12_12=cuCdivcd(crp1[index],pp->b2qbv2);
                    //c1p13_12=crp11[index]/pp.b2qbv3;
                    c1p13_12=cuCdivcd(crp11[index],pp->b2qbv3);
                    //fCF[index][i]=pp.w1p12_2[i]*c1p12_12+pp.w1p13_2[i]*c1p13_12;
                    fCF[index][i]=hipCadd( cuCmuldc(pp->w1p12_2[i],c1p12_12) , cuCmuldc(pp->w1p13_2[i],c1p13_12) );
                
                    break;
                case 13:
                    //1+_3 contribution
                    //c1p12_13=crp1[index]/pp.b2qjv2;
                    c1p12_13=cuCdivcd(crp1[index],pp->b2qjv2);
                    //c1p13_13=crp11[index]/pp.b2qjv3;
                    c1p13_13=cuCdivcd(crp11[index],pp->b2qjv3);
                    //fCF[index][i]=pp.w1p12_3[i]*c1p12_13+pp.w1p13_3[i]*c1p13_13;
                    fCF[index][i]=hipCadd( cuCmuldc(pp->w1p12_3[i],c1p12_13) , cuCmuldc(pp->w1p13_3[i],c1p13_13) );

                    break;
                case 14:
                    //1+_4 contribution
                    //c1p12_12=crp1[index]/pp.b2qbv2;
                    c1p12_12=cuCdivcd(crp1[index],pp->b2qbv2);
                    
                    c1p13_12=cuCdivcd(crp11[index],pp->b2qbv3);
                    c1p12_14=cuCdivcd(c1p12_12,pp->b2qjv2);
                    c1p13_14=cuCdivcd(c1p13_12,pp->b2qjv3);
                    fCF[index][i]=hipCadd( cuCmuldc(pp->w1p12_4[i],c1p12_14), cuCmuldc(pp->w1p13_4[i],c1p13_14));

                    break;
                case 111:
                    //1-__1 contribution
                    cr1m12_1=cuCdivcd( cuCdivcd(crp1[index],pp->b1qjv2) , pp->b1qbv2);
                    cr1m13_1=cuCdivcd( cuCdivcd(crp11[index],pp->b1qjv3) , pp->b1qbv3);
                    fCF[index][i]=hipCadd( cuCmuldc(pp->w1m12[i],cr1m12_1), cuCmuldc(pp->w1m13[i],cr1m13_1));

                    break;
                case 191:
                    //phi(1650)f0(980)_1 contribution
                    //		//cout<<"b1q2r23="<<b1q2r23<<endl;
                    crpf1=cuCdivcd( hipCmul(crp1[index],crp11[index]),pp->b1q2r23 );
                    //		//cout<<"crpf1="<<crpf1<<endl;
                    fCF[index][i]=cuCmuldc(pp->ak23w[i],crpf1);
                    //	//cout<<"fCF[index][i]="<<fCF[index][i]<<endl;

                    break;
                case 192:
                    //phi(1650)f0(980)_2 contribution
                    crpf1=cuCdivcd( hipCmul(crp1[index],crp11[index]) , pp->b1q2r23);
                    crpf2=cuCdivcd(crpf1,pp->b2qjvf2);
                    fCF[index][i]=cuCmuldc(pp->wpf22[i],crpf2);

                    break;
                case 1:
                    //  //cout<<"haha: "<< __LINE__ << endl;
                    //01 contribution
                    //	//cout<<"wu[i]="<<wu[i]<<endl;
                    //	//cout<<"crp1[index]="<<crp1[index]<<endl;
                    //	//cout<<"index="<<index<<endl;
                    fCF[index][i]=cuCmuldc(pp->wu[i],crp1[index]);
                    //	//cout<<"fCF[index][i]="<<fCF[index][i]<<endl;
                    //	//cout<<"i="<<i<<endl;

                    break;
                case 2:
                    //02 contribution
                    cr0p11=cuCdivcd(crp1[index],pp->b2qjvf2);
                    fCF[index][i]=cuCmuldc(pp->w0p22[i],cr0p11);
                    //	//cout<<"fCF[index][i]02="<<fCF[index][i]<<endl;

                    break;
                case 21:
                    //21 contribution
                    //	//cout<<"b2qf2xx="<<b2qf2xx<<endl;
                    cw2p11=cuCdivcd(crp1[index],pp->b2qf2xx);
                    //	//cout<<"cw2p11="<<cw2p11<<endl;
                    //	//cout<<"w2p1[0]="<<w2p1[0]<<endl;
                    //	//cout<<"w2p1[1]="<<w2p1[1]<<endl;
                    fCF[index][i]=cuCmuldc(pp->w2p1[i],cw2p11);
                    //	//cout<<"fCF[index][i]21="<<fCF[index][i]<<endl;

                    break;
                case 22:
                    //22 contribution
                    cw2p11=cuCdivcd(crp1[index],pp->b2qf2xx);
                    cw2p12=cuCdivcd(cw2p11,pp->b2qjvf2);
                    fCF[index][i]=cuCmuldc(pp->w2p2[i],cw2p12);

                    break;
                case 23:
                    //23 contribution
                    cw2p11=cuCdivcd(crp1[index],pp->b2qf2xx);
                    cw2p12=cuCdivcd(cw2p11,pp->b2qjvf2);
                    fCF[index][i]=cuCmuldc(pp->w2p3[i],cw2p12);

                    break;
                case 24:
                    //24 contribution
                    cw2p11=cuCdivcd(crp1[index],pp->b2qf2xx);
                    cw2p12=cuCdivcd(cw2p11,pp->b2qjvf2);
                    fCF[index][i]=cuCmuldc(pp->w2p4[i],cw2p12);

                    break;
                case 25:
                    //25 contribution
                    cw2p11=cuCdivcd(crp1[index],pp->b2qf2xx);
                    cw2p15=cuCdivcd(cw2p11,pp->b4qjvf2);
                    fCF[index][i]=cuCmuldc(pp->w2p5[i],cw2p15);

                default:		;
            }
        }

    }
    double carry(0);
    //#pragmaint host_store_fx(double *h_float_pp,int *h_parameter,double *h_paraList,int para_size, double *h_fx,int numElements,int begin) omp parallel for reduction(+:value)
    for(int i=0;i<const_nAmps;i++){
        //  //cout<<"haha: "<< __LINE__ << endl;    int mlk_cro_size=sizeof(double)*numElements
        for(int j=0;j<const_nAmps;j++){
            cw=hipCmul(fCP[i],hipConj(fCP[j]));
            //    //cout<<"cw="<<cw<<endl;
            if(i==j) pa[i][j]=make_hipDoubleComplex(hipCreal(cw),0.0);
            else if(i<j) pa[i][j]=make_hipDoubleComplex(2*hipCreal(cw),0.0);
            else pa[i][j]=make_hipDoubleComplex(0.0,2*hipCimag(cw));
            cw=make_hipDoubleComplex(0.0,0.0);
            for(int k=0;k<2;k++){
                cw=hipCadd(cw,cuCdivcd( hipCmul( fCF[i][k],hipConj(fCF[j][k]) ),(double)2.0) );
                //   //cout<<"cwfu="<<cw<<endl;

            }
            if(i<=j) fu[i][j]=make_hipDoubleComplex(hipCreal(cw),0.0);
            if(i>j) fu[i][j]=make_hipDoubleComplex(0.0,-hipCimag(cw));
            //      //cout<<"pa[i][j]="<<pa[i][j]<<endl;
            //      //cout<<"fu[i][j]="<<fu[i][j]<<endl;
            double temp = hipCreal( hipCmul(pa[i][j],fu[i][j]) );//i have a big change here 
            double y = temp - carry;
            double t = value + y;
            carry = (t - value) - y;

            value = t; // Kahan Summation
        }
    }

    for(int i=0;i<const_nAmps;i++){
        double2 cw=hipCmul(fCP[i],hipConj(fCP[i]));
        double pa=hipCreal(cw);

        cw=make_hipDoubleComplex(0.0,0.0);
        for(int k=0;k<2;k++){
            //cw+=fCF[i][k]*hipConj(fCF[i][k])/(double)2.0;
            cw=hipCadd(cw,cuCdivcd( hipCmul( fCF[i][k],hipConj(fCF[i][k]) ),(double)2.0) );
        }
        double fu=hipCreal(cw);
        d_mlk[idp*const_nAmps+i] = pa * fu;
    }
    return (value <= 0) ? 1e-20 : value;
}

__global__ void kernel_store_fx(const double * float_pp,const int *parameter,const double *d_paraList,double * d_fx,double *d_mlk,int numElements,int begin)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<numElements && i>= begin)
    {
        int pwa_paras_size = sizeof(cu_PWA_PARAS) / sizeof(double);
        cu_PWA_PARAS *pp = (cu_PWA_PARAS*)&float_pp[i*pwa_paras_size];
        d_fx[i]=calEva(pp,parameter,d_paraList,d_mlk,i);
        printf("\nfx[%d]:%f\n",i,d_fx[i]);
        //fx[i]=calEva(pp,parameter,d_paraList,i);
    }
    
    if(i==1)
    {
        printf("pp[0]:%f pp[end]:%f parameter[0]:%d parameter[16]:%d paraList[0]:%f \n",float_pp[0],float_pp[numElements*sizeof(cu_PWA_PARAS)/sizeof(double)-1],parameter[0],parameter[16],d_paraList[0]);
    }
}

int host_store_fx(double *h_float_pp,int *h_parameter,double *h_paraList,int para_size, double *h_fx,double * h_mlk,int numElements,int begin)
{
    int array_size = sizeof(cu_PWA_PARAS) / sizeof(double) * numElements;
    int mem_size = array_size * sizeof(double);
    //std::cout << __LINE__ << endl;
    double *d_float_pp;
    CUDA_CALL(hipMalloc((void **)&d_float_pp, mem_size));
    CUDA_CALL(hipMemcpy(d_float_pp , h_float_pp, mem_size, hipMemcpyHostToDevice));
    cout << "\nd_float_pp[end]" <<h_float_pp[array_size-1] << endl;
    //std::cout << __LINE__ << endl;
    double *d_fx;
    CUDA_CALL(hipMalloc((void **)&(d_fx),numElements * sizeof(double)));
    //std::cout << __LINE__ << endl;
    int *d_parameter;
    CUDA_CALL(hipMalloc((void **)&(d_parameter),18 * sizeof(int)));
    CUDA_CALL(hipMemcpy(d_parameter , h_parameter, 18*sizeof(int), hipMemcpyHostToDevice));
    cout << "\nd_parameter[16]" <<h_parameter[16] << endl;
    //std::cout << __LINE__ << endl;
    //std::cout << "d_paralist[0]: "<< h_paraList[0] << std::endl;
    //std::cout << "paralist[0]: "<< paraList[0] << std::endl;
    double *d_paraList;
    CUDA_CALL(hipMalloc((void **)&(d_paraList),para_size * sizeof(double)));
    CUDA_CALL(hipMemcpy(d_paraList , h_paraList, para_size * sizeof(double), hipMemcpyHostToDevice));
    cout << "\nd_paraList : " <<h_paraList[0] << endl;
    //std::cout << __LINE__ << endl;

    //init mlk
    double *d_mlk=NULL;
    CUDA_CALL(hipMalloc( (void **)&(d_mlk),(h_parameter[16]+h_parameter[17])*h_parameter[15]*sizeof(double) ));
    //ut << "nAmps="<< h_parameter[15] << "iEnd=" << (h_parameter[16]+h_parameter[17]) << endl;
    int threadsPerBlock = 1;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    kernel_store_fx<<<blocksPerGrid, threadsPerBlock>>>(d_float_pp, d_parameter,d_paraList,d_fx,d_mlk, numElements,begin);
     //std::cout << __LINE__ << endl;
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipMemcpy(h_fx , d_fx, numElements * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_mlk , d_mlk, (h_parameter[16]+h_parameter[17])*h_parameter[15]*sizeof(double), hipMemcpyDeviceToHost));
    //CUDA_CALL(hipMemcpy(h_fx , d_fx, numElements * sizeof(double), hipMemcpyDeviceToHost));

    //free memory
    CUDA_CALL(hipFree(d_float_pp));
    CUDA_CALL(hipFree(d_fx));
    CUDA_CALL(hipFree(d_parameter));
    CUDA_CALL(hipFree(d_paraList));
    CUDA_CALL(hipFree(d_mlk));

    //ofstream cout("data_fx_cal");
    //std::cout << __LINE__ << endl;
    //for(int i=begin;i<numElements;i++)
    //{
        //cout << h_fx[i] << endl;
    //}
    //cout.close();
    return 0;
}


#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"
#include <iostream>
#include "cu_PWA_PARAS.h"
#include <vector>
#include <fstream>
#include <math.h>
#include "cu_DPFPropogator.h"
#include "kernel_calEva.h"
#include <assert.h>
#include <vector>
#include "MultDevice.h"
using namespace std;

//将任何cuda函数作为CUDA_CALL的参数，能够显示返回的错误，并定位错误。 
#define CUDA_CALL(x) {const hipError_t a=(x); if(a != hipSuccess) {printf("\nerror in line:%d CUDAError:%s(err_num=%d)\n",__LINE__,hipGetErrorString(a),a); hipDeviceReset(); assert(0); }}
//block_size 的设定要考虑shared memory 的大小
//shared memory per block ：41952 bytes
//每个block所使用的共享空间大小 ：18×int + paraList + 72×BLOCK_SIZE  
//控制BLOCK_SIZE 使所使用的shared memory 不可高于上限
#define BLOCK_SIZE 64


//calEva是在gpu中运行的一个子程序
 __device__ double calEva(const cu_PWA_PARAS *pp, const int * parameter , double2 * complex_para ,const double * d_paraList,double *d_mlk,int idp) 
    ////return square of complex amplitude
{
    //	static int A=0;
    //	A++;
    
    int _N_spinList     =parameter[0];
    int _N_massList     =parameter[1];
    int _N_mass2List    =parameter[2];
    int _N_widthList    =parameter[3];
    int _N_g1List       =parameter[4];
    int _N_g2List       =parameter[5];
    int _N_b1List       =parameter[6];
    int _N_b2List       =parameter[7];
    int _N_b3List       =parameter[8];
    int _N_b4List       =parameter[9];
    int _N_b5List       =parameter[10];
    int _N_rhoList      =parameter[11];
    int _N_fracList     =parameter[12];
    int _N_phiList      =parameter[13];
    int _N_propList     =parameter[14];
    const int const_nAmps=parameter[15];
    double value = 0.;
    //double2 fCF[const_nAmps][4];
    double2 *fCF=complex_para; 
    //double2 (*fCF)[4]=(double2 (*)[4])malloc(sizeof(double2)*const_nAmps*4);
    //double2 fCP[const_nAmps];
    //double2 * fCP=(double2 *)malloc(sizeof(double2)*const_nAmps);
    double2 * fCP=&complex_para[4*const_nAmps];
    //double2 * crp1=&complex_para[5*const_nAmps];
    //double2 * crp11=&complex_para[6*const_nAmps];


    //double2 pa[const_nAmps][const_nAmps];
    //double2 * pa=&complex_para[7*const_nAmps];
    //double2 * fu=&complex_para[(7+const_nAmps)*const_nAmps];


    /*double2 **pa,**fu;
    pa=(double2 **)malloc(sizeof(double2 *)*const_nAmps);
    fu=(double2 **)malloc(sizeof(double2 *)*const_nAmps);
    for(int i=0;i<const_nAmps;i++)
    {
        pa[i]=(double2 *)malloc(sizeof(double2)*const_nAmps);
        fu[i]=(double2 *)malloc(sizeof(double2)*const_nAmps);
    }
    //double2 fu[const_nAmps][const_nAmps];
    //double2 crp1[const_nAmps];
    double2 * crp1=(double2 *)malloc(sizeof(double2)*const_nAmps);
    //double2 crp11[const_nAmps];
    double2 * crp11=(double2 *)malloc(sizeof(double2)*const_nAmps);
    */
    double2 cr0p11;
    //double2 ca2p1;
    double2 cw2p11;
    double2 cw2p12;
    double2 cw2p15;
    double2 cw;
    double2 c1p12_12,c1p13_12,c1p12_13,c1p13_13,c1p12_14,c1p13_14;
    double2 cr1m12_1,cr1m13_1;
    double2 crpf1,crpf2;

    for(int index=0; index<const_nAmps; index++) {
        double rho0 = d_paraList[_N_rhoList++];
        double frac0 = d_paraList[_N_fracList++];
        double phi0 = d_paraList[_N_phiList++];
        int spin_now = d_paraList[_N_spinList++];
        int propType_now = d_paraList[_N_propList++];
    //cout<<"haha: "<< __LINE__ << endl;
        double2 crp1,crp11;
        rho0 *= std::exp(frac0);
        fCP[index]=make_hipDoubleComplex(rho0*std::cos(phi0),rho0*std::sin(phi0));
        //        //cout<<"fCP[index]="<<fCP[index]<<endl;
        //std::cout << __FILE__ << __LINE__ << " : " << propType_now << std::endl;
        switch(propType_now)
        {
         //  //cout<<"haha: "<< __LINE__ << endl;
            //                     ordinary  Propagator  Contribution
            case 1:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    double mass0 = d_paraList[_N_massList++];
                    double width0 = d_paraList[_N_widthList++];
                    //					//cout<<"mass0="<<mass0<<endl;
                    //					//cout<<"width0="<<width0<<endl;
                    crp1=propogator(mass0,width0,pp->s23);
                }
                break;
            //	Flatte   Propagator Contribution
            case 2:
                {
                    //RooRealVar *g1 = (RooRealVar*)_g1IterV[omp_id]->Next();
                    //RooRealVar *g2 = (RooRealVar*)_g2IterV[omp_id]->Next();
                    double mass980 = d_paraList[_N_massList++];
                    double g10 = d_paraList[_N_g1List++];
                    double g20 = d_paraList[_N_g2List++];
                    //double g10=g1->getVal();
                    //double g20=g2->getVal();
     //               			//cout<<"mass980="<<mass980<<endl;
     //               			//cout<<"g10="<<g10<<endl;
     //               			//cout<<"g20="<<g20<<endl;
     //                           //cout<<"pp.s23="<<pp.s23<< endl;
                    crp1=propogator980(mass980,g10,g20,pp->s23);
     //               			//cout<<"crp1="<<crp1<<endl;
                }
                break;
                // sigma  Propagator Contribution
            case 3:
                {
                    //RooRealVar *b1 = (RooRealVar*)_b1IterV[omp_id]->Next();
                    //RooRealVar *b2 = (RooRealVar*)_b2IterV[omp_id]->Next();
                    //RooRealVar *b3 = (RooRealVar*)_b3IterV[omp_id]->Next();
                    //RooRealVar *b4 = (RooRealVar*)_b4IterV[omp_id]->Next();
                    //RooRealVar *b5 = (RooRealVar*)_b5IterV[omp_id]->Next();
                    //double mass600=mass->getVal();
                    //double b10=b1->getVal();
                    //double b20=b2->getVal();
                    //double b30=b3->getVal();
                    //double b40=b4->getVal();
                    //double b50=b5->getVal();
                    double mass600 = d_paraList[_N_massList++];
                    double b10 = d_paraList[_N_b1List++];
                    double b20 = d_paraList[_N_b2List++];
                    double b30 = d_paraList[_N_b3List++];
                    double b40 = d_paraList[_N_b4List++];
                    double b50 = d_paraList[_N_b5List++];
                    crp1=propogator600(mass600,b10,b20,b30,b40,b50,pp->s23);
                    //			//cout<<"crp13="<<crp1<<endl;
                }
                break;
                // 1- or 1+  Contribution
            case 4:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //double mass0=mass->getVal();
                    //double width0=width->getVal();
                    double mass0 = d_paraList[_N_massList++];
                    double width0 = d_paraList[_N_widthList++];
                    crp1=propogator(mass0,width0,pp->sv2);
                    crp11=propogator(mass0,width0,pp->sv3);
                }
                break;
                //  phi(1650) f0(980) include flatte and ordinary Propagator joint Contribution
            case 5:
                {
                    //RooRealVar *mass2  = (RooRealVar*)_mass2IterV[omp_id]->Next();
                    //RooRealVar *g1 = (RooRealVar*)_g1IterV[omp_id]->Next();
                    //RooRealVar *g2 = (RooRealVar*)_g2IterV[omp_id]->Next();
                    //double mass980=mass2->getVal();
                    //double g10=g1->getVal();
                    //double g20=g2->getVal();
                    double mass980 = d_paraList[_N_mass2List++];
                    double g10 = d_paraList[_N_g1List++];
                    double g20 = d_paraList[_N_g2List++];
                    //					//cout<<"mass980="<<mass980<<endl;
                    //					//cout<<"g10="<<g10<<endl;
                    //					//cout<<"g20="<<g20<<endl;
                    crp1=propogator980(mass980,g10,g20,pp->sv);
                    //					//cout<<"crp1="<<crp1<<endl;
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //double mass1680=mass->getVal();
                    //double width1680=width->getVal();
                    double mass1680 = d_paraList[_N_massList++];
                    double width1680 = d_paraList[_N_widthList++];
                    //					//cout<<"mass1680="<<mass1680<<endl;
                    //					//cout<<"width1680="<<width1680<<endl;
                    crp11=propogator(mass1680,width1680,pp->s23);
                    //					//cout<<"crp11="<<crp11<<endl;
                }
                break;
            case 6:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //double mass0=mass->getVal();
                    //double width0=width->getVal();
                    double mass0 = d_paraList[_N_massList++];
                    double width0 = d_paraList[_N_widthList++];
                    //					//cout<<"mass0="<<mass0<<endl;
                    //					//cout<<"width0="<<width0<<endl;
                    crp1=propogator1270(mass0,width0,pp->s23);
                    //			//cout<<"crp16="<<crp1<<endl;
                }
            default :
                ;
        }
        //if(idp ==1) printf("crp1 : %f\n",hipCreal(crp1));
    //cout << "LINE: " << __LINE__ << endl;
            //if(idp==413) printf("spin_now : %d\n",spin_now);
        for(int i=0;i<2;i++){
            ////cout<<"haha: "<< __LINE__ << endl;
            //		//cout<<"spin_now="<<spin_now<<endl;
            //(idp==413) printf("spin_now : %d\n",spin_now);
            switch(spin_now)
            {
                case 11:
                    //1+_1 contribution
                    //fCF[index][i]=pp.w1p12_1[i]*crp1+pp.w1p13_1[i]*crp11[i];
                    fCF[index*4+i]=hipCadd( cuCmuldc(pp->w1p12_1[i],crp1),cuCmuldc(pp->w1p13_1[i],crp11) );

                    break;
                case 12:
                    //1+_2 contribution
                    //c1p12_12=crp1/pp.b2qbv2;
                    c1p12_12=cuCdivcd(crp1,pp->b2qbv2);
                    //c1p13_12=crp11/pp.b2qbv3;
                    c1p13_12=cuCdivcd(crp11,pp->b2qbv3);
                    //fCF[index][i]=pp.w1p12_2[i]*c1p12_12+pp.w1p13_2[i]*c1p13_12;
                    fCF[index*4+i]=hipCadd( cuCmuldc(pp->w1p12_2[i],c1p12_12) , cuCmuldc(pp->w1p13_2[i],c1p13_12) );
                
                    break;
                case 13:
                    //1+_3 contribution
                    //c1p12_13=crp1/pp.b2qjv2;
                    c1p12_13=cuCdivcd(crp1,pp->b2qjv2);
                    //c1p13_13=crp11/pp.b2qjv3;
                    c1p13_13=cuCdivcd(crp11,pp->b2qjv3);
                    //fCF[index][i]=pp.w1p12_3[i]*c1p12_13+pp.w1p13_3[i]*c1p13_13;
                    fCF[index*4+i]=hipCadd( cuCmuldc(pp->w1p12_3[i],c1p12_13) , cuCmuldc(pp->w1p13_3[i],c1p13_13) );

                    break;
                case 14:
                    //1+_4 contribution
                    //c1p12_12=crp1/pp.b2qbv2;
                    c1p12_12=cuCdivcd(crp1,pp->b2qbv2);
                    
                    c1p13_12=cuCdivcd(crp11,pp->b2qbv3);
                    c1p12_14=cuCdivcd(c1p12_12,pp->b2qjv2);
                    c1p13_14=cuCdivcd(c1p13_12,pp->b2qjv3);
                    fCF[index*4+i]=hipCadd( cuCmuldc(pp->w1p12_4[i],c1p12_14), cuCmuldc(pp->w1p13_4[i],c1p13_14));

                    break;
                case 111:
                    //1-__1 contribution
                    cr1m12_1=cuCdivcd( cuCdivcd(crp1,pp->b1qjv2) , pp->b1qbv2);
                    cr1m13_1=cuCdivcd( cuCdivcd(crp11,pp->b1qjv3) , pp->b1qbv3);
                    fCF[index*4+i]=hipCadd( cuCmuldc(pp->w1m12[i],cr1m12_1), cuCmuldc(pp->w1m13[i],cr1m13_1));

                    break;
                case 191:
                    //phi(1650)f0(980)_1 contribution
                    //		//cout<<"b1q2r23="<<b1q2r23<<endl;
                    crpf1=cuCdivcd( hipCmul(crp1,crp11),pp->b1q2r23 );
                    //		//cout<<"crpf1="<<crpf1<<endl;
                    fCF[index*4+i]=cuCmuldc(pp->ak23w[i],crpf1);
                    //	//cout<<"fCF[index][i]="<<fCF[index][i]<<endl;

                    break;
                case 192:
                    //phi(1650)f0(980)_2 contribution
                    crpf1=cuCdivcd( hipCmul(crp1,crp11) , pp->b1q2r23);
                    crpf2=cuCdivcd(crpf1,pp->b2qjvf2);
                    fCF[index*4+i]=cuCmuldc(pp->wpf22[i],crpf2);

                    break;
                case 1:
                    //  //cout<<"haha: "<< __LINE__ << endl;
                    //01 contribution
                    //	//cout<<"wu[i]="<<wu[i]<<endl;
                    //	//cout<<"crp1="<<crp1<<endl;
                    //	//cout<<"index="<<index<<endl;
                    fCF[index*4+i]=cuCmuldc(pp->wu[i],crp1);
                    //	//cout<<"fCF[index][i]="<<fCF[index][i]<<endl;
                    //	//cout<<"i="<<i<<endl;

                    break;
                case 2:
                    //02 contribution
                    cr0p11=cuCdivcd(crp1,pp->b2qjvf2);
                    fCF[index*4+i]=cuCmuldc(pp->w0p22[i],cr0p11);
                    //	//cout<<"fCF[index][i]02="<<fCF[index][i]<<endl;

                    break;
                case 21:
                    //21 contribution
                    //	//cout<<"b2qf2xx="<<b2qf2xx<<endl;
                    cw2p11=cuCdivcd(crp1,pp->b2qf2xx);
                    //if(idp==413) printf("crp1 : %.10f b2qf2xx : %.10f ",hipCreal(crp1),pp->b2qf2xx);
                    //	//cout<<"cw2p11="<<cw2p11<<endl;
                    //	//cout<<"w2p1[0]="<<w2p1[0]<<endl;
                    //	//cout<<"w2p1[1]="<<w2p1[1]<<endl;
                    fCF[index*4+i]=cuCmuldc(pp->w2p1[i],cw2p11);
                    //if(idp == 413) printf("cw2p11 = %.10f fcf = %.10f\n",hipCimag(cw2p11),hipCimag(fCF[index*4+i]));
                    //	//cout<<"fCF[index][i]21="<<fCF[index][i]<<endl;

                    break;
                case 22:
                    //22 contribution
                    cw2p11=cuCdivcd(crp1,pp->b2qf2xx);
                    cw2p12=cuCdivcd(cw2p11,pp->b2qjvf2);
                    fCF[index*4+i]=cuCmuldc(pp->w2p2[i],cw2p12);

                    break;
                case 23:
                    //23 contribution
                    cw2p11=cuCdivcd(crp1,pp->b2qf2xx);
                    cw2p12=cuCdivcd(cw2p11,pp->b2qjvf2);
                    fCF[index*4+i]=cuCmuldc(pp->w2p3[i],cw2p12);

                    break;
                case 24:
                    //24 contribution
                    cw2p11=cuCdivcd(crp1,pp->b2qf2xx);
                    cw2p12=cuCdivcd(cw2p11,pp->b2qjvf2);
                    fCF[index*4+i]=cuCmuldc(pp->w2p4[i],cw2p12);

                    break;
                case 25:
                    //25 contribution
                    cw2p11=cuCdivcd(crp1,pp->b2qf2xx);
                    cw2p15=cuCdivcd(cw2p11,pp->b4qjvf2);
                    fCF[index*4+i]=cuCmuldc(pp->w2p5[i],cw2p15);

                default:		;
            }
        }

    }
    double carry(0);
    //#pragmaint  omp parallel for reduction(+:value)
    for(int i=0;i<const_nAmps;i++){
        //  //cout<<"haha: "<< __LINE__ << endl;    int mlk_cro_size=sizeof(double)*end
        for(int j=0;j<const_nAmps;j++){
	        double pa,fu;
            cw=hipCmul(fCP[i],hipConj(fCP[j]));
            //    //cout<<"cw="<<cw<<endl;
            if(i==j) pa=hipCreal(cw);
            else if(i<j) pa=2*hipCreal(cw);
            else pa=2*hipCimag(cw);
            //else pa=make_hipDoubleComplex(0.0,2*hipCimag(cw));
            cw=make_hipDoubleComplex(0.0,0.0);
            for(int k=0;k<2;k++){
                cw=hipCadd(cw,cuCdivcd( hipCmul( fCF[i*4+k],hipConj(fCF[j*4+k]) ),2.0) );
                //   //cout<<"cwfu="<<cw<<endl;

            }
            if(i<=j) fu=hipCreal(cw);
            if(i>j) fu=-hipCimag(cw);
            //if(i>j) fu=make_hipDoubleComplex(0.0,-hipCimag(cw));
            //      //cout<<"pa[i][j]="<<pa[i][j]<<endl;
            //      //cout<<"fu[i][j]="<<fu[i][j]<<endl;
            double temp = pa*fu;//i have a big change here 
            double y = temp - carry;
            double t = value + y;
            carry = (t - value) - y;

            value = t; // Kahan Summation
        }
    }

    for(int i=0;i<const_nAmps;i++){
        double2 cw=hipCmul(fCP[i],hipConj(fCP[i]));
        double pa=hipCreal(cw);

        cw=make_hipDoubleComplex(0.0,0.0);
        for(int k=0;k<2;k++){
            //cw+=fCF[i][k]*hipConj(fCF[i][k])/(double)2.0;
            cw=hipCadd(cw,cuCdivcd( hipCmul( fCF[i*4+k],hipConj(fCF[i*4+k]) ),2.0) );
        }
        double fu=hipCreal(cw);
        d_mlk[idp*const_nAmps+i] = pa * fu;
        //if(idp==413 && i==3 ) printf("pa: %.10f  fu: %.10f mlk %.10f\n",pa ,fu,d_mlk[idp*const_nAmps+i]);
    }
    /*
    free(fCP);
    for(int i=0;i<const_nAmps;i++)
    {
        free(pa[i]);
        free(fu[i]);
        //free(fCF[i]);
    } 
    free(fCF);
    free(pa);
    free(fu);
    free(crp1);
    free(crp11);
*/
    //if(idp==1) printf("%f %d %f \n", pp->wu[0] ,_N_spinList,d_paraList[0]);
    return (value <= 0) ? 1e-20 : value;
}

__global__ void kernel_store_fx(const double * float_pp,const int *parameter,double2 * d_complex_para ,const double *d_paraList,int para_size,double * d_fx,double *d_mlk,int end,int begin)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    //使用shared memory 开辟静态内存
    __shared__ int sh_parameter[18];
    for(int i=0;i<18;i++)
        sh_parameter[i]=parameter[i];
    //使用shared memory 开辟动态内存
    extern __shared__ double sh_paraList[];
    for(int i=0;i<para_size;i++)
        sh_paraList[i]=d_paraList[i];

    if(i<end-begin && i>= 0)
    {
        int pwa_paras_size = sizeof(cu_PWA_PARAS) / sizeof(double);
        //cu_PWA_PARAS * pp= (cu_PWA_PARAS *)&float_pp[i*pwa_paras_size];
        __shared__ double sh_float_pp[BLOCK_SIZE*72];
        const double *pp = &float_pp[(i+begin)*pwa_paras_size];
        for(int j=0;j<72;j++)
        {
            sh_float_pp[threadIdx.x*72+j]=pp[j];
        }
        cu_PWA_PARAS *sh_pp=(cu_PWA_PARAS*)&sh_float_pp[threadIdx.x*72];
        double2 *complex_para=&d_complex_para[i*6*parameter[15]];
        //将各个参数传到gpu中的内存后，调用子函数calEva 
        d_fx[i]=calEva(sh_pp,sh_parameter,complex_para,sh_paraList,d_mlk,i);
        //printf("%dgpu :: %.7f\n",i,pp->wu[0]);
        //printf("\nfx[%d]:%f\n",i,d_fx[i]);
        //fx[i]=calEva(pp,parameter,d_paraList,i);
    }
    //if(i==1)
    //{
        //printf("pp[0]:%f pp[end]:%f parameter[0]:%d parameter[16]:%d paraList[0]:%f \n",float_pp[0],float_pp[end*sizeof(cu_PWA_PARAS)/sizeof(double)-1],parameter[0],parameter[16],d_paraList[0]);
    //}
}

int host_store_fx(vector<double *> d_float_pp,int *h_parameter,double *h_paraList,int para_size, double *h_fx,double * h_mlk,int end,int begin)
{
    //init Ns
    //Ns为分段数组，第i个gpu所处理的线程序号范围为:[ Ns[i] , Ns[i+1] ) 
    int Ns[DEVICE_NUM+1];
    Ns[0]=0;
    for(int i=1;i<DEVICE_NUM;i++)
    {
        Ns[i]=Ns[i-1]+end/DEVICE_NUM;
    }
    Ns[DEVICE_NUM]=end;
    //malloc memory
    double *d_fx[DEVICE_NUM];
    int *d_parameter[DEVICE_NUM];
    double *d_paraList[DEVICE_NUM];
    double2 * d_complex_para[DEVICE_NUM];
    double *d_mlk[DEVICE_NUM];
    for(int i=0;i<DEVICE_NUM;i++)
    {
        CUDA_CALL( hipSetDevice(i) );
        int N_thread=Ns[i+1]-Ns[i];
        CUDA_CALL(hipMalloc((void **)&(d_fx[i]),N_thread * sizeof(double)));
        CUDA_CALL(hipMalloc((void **)&(d_parameter[i]),18 * sizeof(int)));
        CUDA_CALL(hipMalloc((void **)&(d_paraList[i]),para_size * sizeof(double)));
        CUDA_CALL(hipMalloc( (void**)&d_complex_para[i],6*h_parameter[15]*N_thread *sizeof(double2) ));
        CUDA_CALL(hipMalloc( (void **)&(d_mlk[i]),(N_thread*h_parameter[15]*sizeof(double) )));
    }
    //动态分配shared memory 的大小：
    int size_paraList=para_size*sizeof(double);
    //memcpy d_parameter
    for(int i=0;i<DEVICE_NUM;i++)
    {
        CUDA_CALL(hipSetDevice(i) );
        //使用异步函数。
        CUDA_CALL(hipMemcpyAsync(d_parameter[i] , h_parameter, 18*sizeof(int), hipMemcpyHostToDevice));
    }
    //memcpy d_paraList
    for(int i=0;i<DEVICE_NUM;i++)
    {
        CUDA_CALL(hipSetDevice(i) );
        //使用异步函数.
        CUDA_CALL(hipMemcpyAsync(d_paraList[i] , h_paraList, para_size * sizeof(double), hipMemcpyHostToDevice));
    }
    int threadsPerBlock = BLOCK_SIZE;
    for(int i=0;i<DEVICE_NUM;i++)
    {
        CUDA_CALL(hipSetDevice(i) );
        int N_thread=Ns[i+1]-Ns[i];
        int blocksPerGrid =(N_thread + threadsPerBlock - 1) / threadsPerBlock;
        printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        kernel_store_fx<<<blocksPerGrid, threadsPerBlock,size_paraList>>>(d_float_pp[i], d_parameter[i],d_complex_para[i],d_paraList[i],para_size,d_fx[i],d_mlk[i],Ns[i+1],Ns[i]);
    }
    for(int i=0;i<DEVICE_NUM;i++)
    {
        CUDA_CALL(hipSetDevice(i) );
        int N_thread=Ns[i+1]-Ns[i];
        CUDA_CALL(hipMemcpyAsync(&h_fx[Ns[i]] , d_fx[i], N_thread * sizeof(double), hipMemcpyDeviceToHost));
        //CUDA_CALL(hipMemcpyAsync(&h_mlk[ Ns[i]*h_parameter[15] ] , d_mlk[i], N_thread * h_parameter[15]*sizeof(double), hipMemcpyDeviceToHost));
    }
    for(int i=0;i<DEVICE_NUM;i++)
    {
        CUDA_CALL(hipSetDevice(i) );
        int N_thread=Ns[i+1]-Ns[i];
        //CUDA_CALL(hipMemcpyAsync(&h_fx[Ns[i]] , d_fx[i], N_thread * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyAsync(&h_mlk[ Ns[i]*h_parameter[15] ] , d_mlk[i], N_thread * h_parameter[15]*sizeof(double), hipMemcpyDeviceToHost));
    }
    //free memory
    //CUDA_CALL(hipFree(d_float_pp));
    for(int i=0;i<DEVICE_NUM;i++)
    {
        CUDA_CALL(hipSetDevice(i) );
        CUDA_CALL(hipFree(d_fx[i]));
        CUDA_CALL(hipFree(d_complex_para[i]));
        CUDA_CALL(hipFree(d_parameter[i]));
        CUDA_CALL(hipFree(d_paraList[i]));
        CUDA_CALL(hipFree(d_mlk[i]));
    }
    //ofstream cout("data_fx_cal");
    //std::cout << __LINE__ << endl;
    //for(int i=begin;i<end;i++)
    //{
        //cout << h_fx[i] << endl;
    //}
    //cout.close();
    return 0;
}
//在gpu中为pwa_paras开辟空间
void cu_malloc_h_pp(double *h_float_pp,double *&d_float_pp,int length,int device)
{
    CUDA_CALL( hipSetDevice(device) );
    int array_size = sizeof(cu_PWA_PARAS) / sizeof(double) * length;
    int mem_size = array_size * sizeof(double);
    CUDA_CALL(hipMalloc((void **)&d_float_pp, mem_size));
    CUDA_CALL(hipMemcpy(d_float_pp , h_float_pp, mem_size, hipMemcpyHostToDevice));
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"
#include <iostream>
#include "calEva.h"
#include "PWA_PARAS.h"
#include <vector>
#include <fstream>
#include <math.h>
#include "DPFPropogator.h"
#include "kernel_calEva.h"
#include "conf.h"

using namespace std;

#define CUDA_CALL(x) {const hipError_t a=(x); if(a != hipSuccess) {printf("\nCUDAError:%s(err_num=%d)\n",hipGetErrorString(a),a); hipDeviceReset(); }}

    int _CN_spinList;
    int _CN_massList;
    int _CN_mass2List;
    int _CN_widthList;
    int _CN_g1List;
    int _CN_g2List;
    int _CN_b1List;
    int _CN_b2List;
    int _CN_b3List;
    int _CN_b4List;
    int _CN_b5List;
    int _CN_rhoList;
    int _CN_fracList;
    int _CN_phiList;
    int _CN_propList;
    int nAmps;
    int Nmc,Nmc_data;
    std::vector<double> paraList;
    my_float **mlk;

__global__ void
convert(const my_float *A, my_float *BB, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        int pwa_paras_size = sizeof(PWA_PARAS) / sizeof(my_float);
        //my_float *pp = (my_float *)malloc(sizeof(PWA_PARAS));
        //for(int j = 0; j < pwa_paras_size; j++) {
           // pp[j] = A[i * pwa_paras_size + j];
       // }
        //PWA_PARAS tt = ((PWA_PARAS*)pp)[0];
        PWA_PARAS *tt = (PWA_PARAS*)&A[i*pwa_paras_size];
        BB[i] = tt->wu[0] + tt->wu[1] + tt->wu[2] + tt->wu[3];
    }
}


 __device__ my_float calEva(const PWA_PARAS *pp, const int * parameter , const double * d_paraList,int idp) 
    ////return square of complex amplitude
{
    //	static int A=0;
    //	A++;
    
    int _N_spinList     =parameter[0];
    int _N_massList     =parameter[1];
    int _N_mass2List    =parameter[2];
    int _N_widthList    =parameter[3];
    int _N_g1List       =parameter[4];
    int _N_g2List       =parameter[5];
    int _N_b1List       =parameter[6];
    int _N_b2List       =parameter[7];
    int _N_b3List       =parameter[8];
    int _N_b4List       =parameter[9];
    int _N_b5List       =parameter[10];
    int _N_rhoList      =parameter[11];
    int _N_fracList     =parameter[12];
    int _N_phiList      =parameter[13];
    int _N_propList     =parameter[14];
    const int const_nAmps=parameter[15];
    my_float value = 0.;
    //TComplex fCF[const_nAmps][4];
    TComplex (*fCF)[4]=(TComplex (*)[4])malloc(sizeof(TComplex)*const_nAmps*4);
    //TComplex fCP[const_nAmps];
    TComplex * fCP=(TComplex *)malloc(sizeof(TComplex)*const_nAmps);
    //TComplex pa[const_nAmps][const_nAmps];
    TComplex **pa,**fu;
    pa=(TComplex **)malloc(sizeof(TComplex *)*const_nAmps);
    fu=(TComplex **)malloc(sizeof(TComplex *)*const_nAmps);
    for(int i=0;i<const_nAmps;i++)
    {
        pa[i]=(TComplex *)malloc(sizeof(TComplex)*const_nAmps);
        fu[i]=(TComplex *)malloc(sizeof(TComplex)*const_nAmps);
    }
    //TComplex fu[const_nAmps][const_nAmps];
    //TComplex crp1[const_nAmps];
    TComplex * crp1=(TComplex *)malloc(sizeof(TComplex)*const_nAmps);
    //TComplex crp11[const_nAmps];
    TComplex * crp11=(TComplex *)malloc(sizeof(TComplex)*const_nAmps);
    TComplex cr0p11;
    //TComplex ca2p1;
    TComplex cw2p11;
    TComplex cw2p12;
    TComplex cw2p15;
    TComplex cw;
    TComplex c1p12_12,c1p13_12,c1p12_13,c1p13_13,c1p12_14,c1p13_14;
    TComplex cr1m12_1,cr1m13_1;
    TComplex crpf1,crpf2;

    for(int index=0; index<const_nAmps; index++) {
        my_float rho0 = d_paraList[_N_rhoList++];
        my_float frac0 = d_paraList[_N_fracList++];
        my_float phi0 = d_paraList[_N_phiList++];
        int spin_now = d_paraList[_N_spinList++];
        int propType_now = d_paraList[_N_propList++];
    //cout<<"haha: "<< __LINE__ << endl;

        rho0 *= std::exp(frac0);
        fCP[index]=make_complex(rho0*std::cos(phi0),rho0*std::sin(phi0));
        //        //cout<<"fCP[index]="<<fCP[index]<<endl;
        //std::cout << __FILE__ << __LINE__ << " : " << propType_now << std::endl;
        switch(propType_now)
        {
            //  //cout<<"haha: "<< __LINE__ << endl;
            //                     ordinary  Propagator  Contribution
            case 1:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    my_float mass0 = d_paraList[_N_massList++];
                    my_float width0 = d_paraList[_N_widthList++];
                    //					//cout<<"mass0="<<mass0<<endl;
                    //					//cout<<"width0="<<width0<<endl;
                    crp1[index]=propogator(mass0,width0,pp->s23);
                }
                break;
            //	Flatte   Propagator Contribution
            case 2:
                {
                    //RooRealVar *g1 = (RooRealVar*)_g1IterV[omp_id]->Next();
                    //RooRealVar *g2 = (RooRealVar*)_g2IterV[omp_id]->Next();
                    my_float mass980 = d_paraList[_N_massList++];
                    my_float g10 = d_paraList[_N_g1List++];
                    my_float g20 = d_paraList[_N_g2List++];
                    //my_float g10=g1->getVal();
                    //my_float g20=g2->getVal();
     //               			//cout<<"mass980="<<mass980<<endl;
     //               			//cout<<"g10="<<g10<<endl;
     //               			//cout<<"g20="<<g20<<endl;
     //                           //cout<<"pp.s23="<<pp.s23<< endl;
                    crp1[index]=propogator980(mass980,g10,g20,pp->s23);
     //               			//cout<<"crp1[index]="<<crp1[index]<<endl;
                }
                break;
                // sigma  Propagator Contribution
            case 3:
                {
                    //RooRealVar *b1 = (RooRealVar*)_b1IterV[omp_id]->Next();
                    //RooRealVar *b2 = (RooRealVar*)_b2IterV[omp_id]->Next();
                    //RooRealVar *b3 = (RooRealVar*)_b3IterV[omp_id]->Next();
                    //RooRealVar *b4 = (RooRealVar*)_b4IterV[omp_id]->Next();
                    //RooRealVar *b5 = (RooRealVar*)_b5IterV[omp_id]->Next();
                    //my_float mass600=mass->getVal();
                    //my_float b10=b1->getVal();
                    //my_float b20=b2->getVal();
                    //my_float b30=b3->getVal();
                    //my_float b40=b4->getVal();
                    //my_float b50=b5->getVal();
                    my_float mass600 = d_paraList[_N_massList++];
                    my_float b10 = d_paraList[_N_b1List++];
                    my_float b20 = d_paraList[_N_b2List++];
                    my_float b30 = d_paraList[_N_b3List++];
                    my_float b40 = d_paraList[_N_b4List++];
                    my_float b50 = d_paraList[_N_b5List++];
                    crp1[index]=propogator600(mass600,b10,b20,b30,b40,b50,pp->s23);
                    //			//cout<<"crp1[index]3="<<crp1[index]<<endl;
                }
                break;
                // 1- or 1+  Contribution
            case 4:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //my_float mass0=mass->getVal();
                    //my_float width0=width->getVal();
                    my_float mass0 = d_paraList[_N_massList++];
                    my_float width0 = d_paraList[_N_widthList++];
                    crp1[index]=propogator(mass0,width0,pp->sv2);
                    crp11[index]=propogator(mass0,width0,pp->sv3);
                }
                break;
                //  phi(1650) f0(980) include flatte and ordinary Propagator joint Contribution
            case 5:
                {
                    //RooRealVar *mass2  = (RooRealVar*)_mass2IterV[omp_id]->Next();
                    //RooRealVar *g1 = (RooRealVar*)_g1IterV[omp_id]->Next();
                    //RooRealVar *g2 = (RooRealVar*)_g2IterV[omp_id]->Next();
                    //my_float mass980=mass2->getVal();
                    //my_float g10=g1->getVal();
                    //my_float g20=g2->getVal();
                    my_float mass980 = d_paraList[_N_mass2List++];
                    my_float g10 = d_paraList[_N_g1List++];
                    my_float g20 = d_paraList[_N_g2List++];
                    //					//cout<<"mass980="<<mass980<<endl;
                    //					//cout<<"g10="<<g10<<endl;
                    //					//cout<<"g20="<<g20<<endl;
                    crp1[index]=propogator980(mass980,g10,g20,pp->sv);
                    //					//cout<<"crp1[index]="<<crp1[index]<<endl;
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //my_float mass1680=mass->getVal();
                    //my_float width1680=width->getVal();
                    my_float mass1680 = d_paraList[_N_massList++];
                    my_float width1680 = d_paraList[_N_widthList++];
                    //					//cout<<"mass1680="<<mass1680<<endl;
                    //					//cout<<"width1680="<<width1680<<endl;
                    crp11[index]=propogator(mass1680,width1680,pp->s23);
                    //					//cout<<"crp11[index]="<<crp11[index]<<endl;
                }
                break;
            case 6:
                {
                    //RooRealVar *width = (RooRealVar*)_widthIterV[omp_id]->Next();
                    //my_float mass0=mass->getVal();
                    //my_float width0=width->getVal();
                    my_float mass0 = d_paraList[_N_massList++];
                    my_float width0 = d_paraList[_N_widthList++];
                    //					//cout<<"mass0="<<mass0<<endl;
                    //					//cout<<"width0="<<width0<<endl;
                    crp1[index]=propogator1270(mass0,width0,pp->s23);
                    //			//cout<<"crp1[index]6="<<crp1[index]<<endl;
                }
            default :
                ;
        }
    //cout << "LINE: " << __LINE__ << endl;
        for(int i=0;i<2;i++){
            ////cout<<"haha: "<< __LINE__ << endl;
            //		//cout<<"spin_now="<<spin_now<<endl;
            switch(spin_now)
            {
                case 11:
                    //1+_1 contribution
                    //fCF[index][i]=pp.w1p12_1[i]*crp1[index]+pp.w1p13_1[i]*crp11[i];
                    fCF[index][i]=hipCadd( cuCmuldc(pp->w1p12_1[i],crp1[index]),cuCmuldc(pp->w1p13_1[i],crp11[i]) );

                    break;
                case 12:
                    //1+_2 contribution
                    //c1p12_12=crp1[index]/pp.b2qbv2;
                    c1p12_12=cuCdivcd(crp1[index],pp->b2qbv2);
                    //c1p13_12=crp11[index]/pp.b2qbv3;
                    c1p13_12=cuCdivcd(crp11[index],pp->b2qbv3);
                    //fCF[index][i]=pp.w1p12_2[i]*c1p12_12+pp.w1p13_2[i]*c1p13_12;
                    fCF[index][i]=hipCadd( cuCmuldc(pp->w1p12_2[i],c1p12_12) , cuCmuldc(pp->w1p13_2[i],c1p13_12) );
                
                    break;
                case 13:
                    //1+_3 contribution
                    //c1p12_13=crp1[index]/pp.b2qjv2;
                    c1p12_13=cuCdivcd(crp1[index],pp->b2qjv2);
                    //c1p13_13=crp11[index]/pp.b2qjv3;
                    c1p13_13=cuCdivcd(crp11[index],pp->b2qjv3);
                    //fCF[index][i]=pp.w1p12_3[i]*c1p12_13+pp.w1p13_3[i]*c1p13_13;
                    fCF[index][i]=hipCadd( cuCmuldc(pp->w1p12_3[i],c1p12_13) , cuCmuldc(pp->w1p13_3[i],c1p13_13) );

                    break;
                case 14:
                    //1+_4 contribution
                    //c1p12_12=crp1[index]/pp.b2qbv2;
                    c1p12_12=cuCdivcd(crp1[index],pp->b2qbv2);
                    
                    c1p13_12=cuCdivcd(crp11[index],pp->b2qbv3);
                    c1p12_14=cuCdivcd(c1p12_12,pp->b2qjv2);
                    c1p13_14=cuCdivcd(c1p13_12,pp->b2qjv3);
                    fCF[index][i]=hipCadd( cuCmuldc(pp->w1p12_4[i],c1p12_14), cuCmuldc(pp->w1p13_4[i],c1p13_14));

                    break;
                case 111:
                    //1-__1 contribution
                    cr1m12_1=cuCdivcd( cuCdivcd(crp1[index],pp->b1qjv2) , pp->b1qbv2);
                    cr1m13_1=cuCdivcd( cuCdivcd(crp11[index],pp->b1qjv3) , pp->b1qbv3);
                    fCF[index][i]=hipCadd( cuCmuldc(pp->w1m12[i],cr1m12_1), cuCmuldc(pp->w1m13[i],cr1m13_1));

                    break;
                case 191:
                    //phi(1650)f0(980)_1 contribution
                    //		//cout<<"b1q2r23="<<b1q2r23<<endl;
                    crpf1=cuCdivcd( hipCmul(crp1[index],crp11[index]),pp->b1q2r23 );
                    //		//cout<<"crpf1="<<crpf1<<endl;
                    fCF[index][i]=cuCmuldc(pp->ak23w[i],crpf1);
                    //	//cout<<"fCF[index][i]="<<fCF[index][i]<<endl;

                    break;
                case 192:
                    //phi(1650)f0(980)_2 contribution
                    crpf1=cuCdivcd( hipCmul(crp1[index],crp11[index]) , pp->b1q2r23);
                    crpf2=cuCdivcd(crpf1,pp->b2qjvf2);
                    fCF[index][i]=cuCmuldc(pp->wpf22[i],crpf2);

                    break;
                case 1:
                    //  //cout<<"haha: "<< __LINE__ << endl;
                    //01 contribution
                    //	//cout<<"wu[i]="<<wu[i]<<endl;
                    //	//cout<<"crp1[index]="<<crp1[index]<<endl;
                    //	//cout<<"index="<<index<<endl;
                    fCF[index][i]=cuCmuldc(pp->wu[i],crp1[index]);
                    //	//cout<<"fCF[index][i]="<<fCF[index][i]<<endl;
                    //	//cout<<"i="<<i<<endl;

                    break;
                case 2:
                    //02 contribution
                    cr0p11=cuCdivcd(crp1[index],pp->b2qjvf2);
                    fCF[index][i]=cuCmuldc(pp->w0p22[i],cr0p11);
                    //	//cout<<"fCF[index][i]02="<<fCF[index][i]<<endl;

                    break;
                case 21:
                    //21 contribution
                    //	//cout<<"b2qf2xx="<<b2qf2xx<<endl;
                    cw2p11=cuCdivcd(crp1[index],pp->b2qf2xx);
                    //	//cout<<"cw2p11="<<cw2p11<<endl;
                    //	//cout<<"w2p1[0]="<<w2p1[0]<<endl;
                    //	//cout<<"w2p1[1]="<<w2p1[1]<<endl;
                    fCF[index][i]=cuCmuldc(pp->w2p1[i],cw2p11);
                    //	//cout<<"fCF[index][i]21="<<fCF[index][i]<<endl;

                    break;
                case 22:
                    //22 contribution
                    cw2p11=cuCdivcd(crp1[index],pp->b2qf2xx);
                    cw2p12=cuCdivcd(cw2p11,pp->b2qjvf2);
                    fCF[index][i]=cuCmuldc(pp->w2p2[i],cw2p12);

                    break;
                case 23:
                    //23 contribution
                    cw2p11=cuCdivcd(crp1[index],pp->b2qf2xx);
                    cw2p12=cuCdivcd(cw2p11,pp->b2qjvf2);
                    fCF[index][i]=cuCmuldc(pp->w2p3[i],cw2p12);

                    break;
                case 24:
                    //24 contribution
                    cw2p11=cuCdivcd(crp1[index],pp->b2qf2xx);
                    cw2p12=cuCdivcd(cw2p11,pp->b2qjvf2);
                    fCF[index][i]=cuCmuldc(pp->w2p4[i],cw2p12);

                    break;
                case 25:
                    //25 contribution
                    cw2p11=cuCdivcd(crp1[index],pp->b2qf2xx);
                    cw2p15=cuCdivcd(cw2p11,pp->b4qjvf2);
                    fCF[index][i]=cuCmuldc(pp->w2p5[i],cw2p15);

                default:		;
            }
        }

    }
    my_float carry(0);
    //#pragma omp parallel for reduction(+:value)
    for(int i=0;i<const_nAmps;i++){
        //  //cout<<"haha: "<< __LINE__ << endl;
        for(int j=0;j<const_nAmps;j++){
            cw=hipCmul(fCP[i],hipConj(fCP[j]));
            //    //cout<<"cw="<<cw<<endl;
            if(i==j) pa[i][j]=make_complex(hipCreal(cw),0.0);
            else if(i<j) pa[i][j]=make_complex(2*hipCreal(cw),0.0);
            else pa[i][j]=make_complex(0.0,2*hipCimag(cw));
            cw=make_complex(0.0,0.0);
            for(int k=0;k<2;k++){
                cw=hipCadd(cw,cuCdivcd( hipCmul( fCF[i][k],hipConj(fCF[j][k]) ),(my_float)2.0) );
                //   //cout<<"cwfu="<<cw<<endl;

            }
            if(i<=j) fu[i][j]=make_complex(hipCreal(cw),0.0);
            if(i>j) fu[i][j]=make_complex(0.0,-hipCimag(cw));
            //      //cout<<"pa[i][j]="<<pa[i][j]<<endl;
            //      //cout<<"fu[i][j]="<<fu[i][j]<<endl;
            my_float temp = hipCreal( hipCmul(pa[i][j],fu[i][j]) );//i have a big change here 
            my_float y = temp - carry;
            my_float t = value + y;
            carry = (t - value) - y;

            value = t; // Kahan Summation
        }
    }

    for(int i=0;i<const_nAmps;i++){
        TComplex cw=hipCmul(fCP[i],hipConj(fCP[i]));
        my_float pa=hipCreal(cw);

        cw=make_complex(0.0,0.0);
        for(int k=0;k<2;k++){
            //cw+=fCF[i][k]*hipConj(fCF[i][k])/(my_float)2.0;
            cw=hipCadd(cw,cuCdivcd( hipCmul( fCF[i][k],hipConj(fCF[i][k]) ),(my_float)2.0) );
        }
        my_float fu=hipCreal(cw);
       // mlk[idp][i] = pa * fu;
    }
    return (value <= 0) ? 1e-20 : value;
}

__global__ void kernel_store_fx(const my_float * float_pp,const int *parameter,const double *d_paraList,my_float * d_fx,int numElements,int begin)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<numElements && i>= begin)
    {
        int pwa_paras_size = sizeof(PWA_PARAS) / sizeof(my_float);
        PWA_PARAS *pp = (PWA_PARAS*)&float_pp[i*pwa_paras_size];
        d_fx[i]=calEva(pp,parameter,d_paraList,i);
    }
    //if(i==1)
    //{
    //    printf("pp[0]:%f pp[end]:%f parameter[0]:%d parameter[17]:%d paraList[0]:%f \n",float_pp[0],float_pp[numElements*sizeof(PWA_PARAS)/sizeof(my_float)],parameter[0],parameter[17],d_paraList[0]);
    //}
}

int host_store_fx(my_float *h_float_pp,int *h_parameter,double *h_paraList,int para_size, my_float *h_fx,int numElements,int begin)
{
    int array_size = sizeof(PWA_PARAS) / sizeof(my_float) * iEnd;
    int mem_size = array_size * sizeof(my_float);
    //std::cout << __LINE__ << endl;
    my_float *d_float_pp;
    CUDA_CALL(hipMalloc((void **)&d_float_pp, mem_size));
    CUDA_CALL(hipMemcpy(d_float_pp , h_float_pp, mem_size, hipMemcpyHostToDevice));
     //std::cout << __LINE__ << endl;
    my_float *d_fx;
    CUDA_CALL(hipMalloc((void **)&(d_fx),numElements * sizeof(my_float)));
     //std::cout << __LINE__ << endl;
    int *d_parameter;
    CUDA_CALL(hipMalloc((void **)&(d_parameter),18 * sizeof(int)));
    CUDA_CALL(hipMemcpy(d_parameter , h_parameter, 18*sizeof(int), hipMemcpyHostToDevice));
     //std::cout << __LINE__ << endl;
    //std::cout << "d_paralist[0]: "<< h_paraList[0] << std::endl;
    //std::cout << "paralist[0]: "<< paraList[0] << std::endl;
    double *d_paraList;
    CUDA_CALL(hipMalloc((void **)&(d_paraList),para_size * sizeof(double)));
    CUDA_CALL(hipMemcpy(d_paraList , h_paraList, para_size * sizeof(double), hipMemcpyHostToDevice));
     //std::cout << __LINE__ << endl;
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    kernel_store_fx<<<blocksPerGrid, threadsPerBlock>>>(d_float_pp, d_parameter,d_paraList,d_fx, numElements,begin);
     //std::cout << __LINE__ << endl;
    h_fx[0]=0;
    CUDA_CALL(hipMemcpy(h_fx , d_fx, numElements * sizeof(my_float), hipMemcpyDeviceToHost));
    ofstream cout("data_fx_cal");
    //std::cout << __LINE__ << endl;
    for(int i=begin;i<numElements;i++)
    {
        cout << h_fx[i] << endl;
    }
    cout.close();
    return 0;
}

void func(DataPointers& cpu_data_pointers)
{
    my_float * h_float_pp=cpu_data_pointers.pointer_data;

    int * h_parameter=(int *)malloc(18*sizeof(int));
    h_parameter[0] =  _CN_spinList;
    h_parameter[1] =  _CN_massList;
    h_parameter[2] =  _CN_mass2List;
    h_parameter[3] =  _CN_widthList;
    h_parameter[4] =  _CN_g1List;
    h_parameter[5] =  _CN_g2List;
    h_parameter[6] =  _CN_b1List;
    h_parameter[7] =  _CN_b2List;
    h_parameter[8] =  _CN_b3List;
    h_parameter[9] =  _CN_b4List;
    h_parameter[10] =  _CN_b5List;
    h_parameter[11] =  _CN_rhoList;
    h_parameter[12] =  _CN_fracList;
    h_parameter[13] =  _CN_phiList;
    h_parameter[14] =  _CN_propList;
    h_parameter[15] =  nAmps;
    h_parameter[16] =  Nmc;
    h_parameter[17] = Nmc_data; 

    double * h_paraList=(double *)malloc(paraList.size()*sizeof(double));
    for(int i=0;i<paraList.size();i++)
    {
        h_paraList[i]=paraList[i];
    }
    
    my_float *h_fx=(my_float *)malloc(iEnd*sizeof(my_float));

    host_store_fx(h_float_pp,h_parameter,h_paraList,paraList.size(),h_fx,iEnd,iBegin);
}
//将文件中的数据pwa_paras读出来，存在数组中，内存中的存储是一定的。但是结构题的指针可以随意转化
int initialize_data(std::vector<PWA_PARAS> &pwa_paras, DataPointers& cpu_data_points)
{
    mlk = new my_float*[Nmc + Nmc_data];
    for(int i = 0; i < Nmc + Nmc_data; i++) {
        mlk[i] = new my_float[nAmps];
    }
    //init mlk
    //init private num
    std::fstream cin("data_of_private_member");
         cin >> _CN_spinList ;
        cin >>  _CN_massList ;
        cin >>  _CN_mass2List ;
        cin >>  _CN_widthList ;
        cin >>  _CN_g1List ;
        cin >>  _CN_g2List ;
        cin >>  _CN_b1List ;
        cin >>  _CN_b2List ;
        cin >>  _CN_b3List ;
        cin >>  _CN_b4List ;
        cin >>  _CN_b5List ;
        cin >>  _CN_rhoList ;
        cin >>  _CN_fracList ;
        cin >>  _CN_phiList ;
        cin >>  _CN_propList ;
        cin >>  nAmps ;
        cin >>  Nmc ;
        cin >> Nmc_data ;
        int paraList_size;
        cin >> paraList_size;
        paraList.resize(paraList_size);
        for(int i=0;i<paraList_size;i++)
        {
            cin >> paraList[i] ;
        }
        cin.close();
    ///////////////////////////////
    pwa_paras.resize(iEnd);
    int array_size = sizeof(PWA_PARAS) / sizeof(my_float) * iEnd;
    int mem_size = array_size * sizeof(my_float);
    std::cout << "array_size=" << array_size << std::endl;

    cpu_data_points.pointer_data = (my_float *)malloc(mem_size);
    cpu_data_points.pointer_data_pwa_paras_type = (PWA_PARAS*)cpu_data_points.pointer_data;

    cpu_data_points.result_data = (my_float*)malloc(array_size);
    std::cout << "finish cpu memory malloc" << std::endl;
    std::ifstream in("data_pwa_paras");
    my_float temp_num;
    for(int i = 0; i < array_size; i++) {
        in >> temp_num;
        cpu_data_points.pointer_data[i] = temp_num;
    }
    in.close();
std::cout << "haha" << __LINE__ << std::endl;
    for(int i = 0; i < iEnd; i++) {
        pwa_paras[i] = cpu_data_points.pointer_data_pwa_paras_type[i];
    }
    return 0;
}

int data_distribution(DataPointers& cpu_data_pointers, CudaDataPointers& cuda_data_pointers)
{
    int array_size = sizeof(PWA_PARAS) / sizeof(my_float) * iEnd;
    int mem_size = array_size * sizeof(my_float);
    CUDA_CALL(hipMalloc((void **)&(cuda_data_pointers.input_data), mem_size));
    CUDA_CALL(hipMalloc((void **)&(cuda_data_pointers.output_data), iEnd * sizeof(my_float)));
    CUDA_CALL(hipMemcpy(cuda_data_pointers.input_data, cpu_data_pointers.pointer_data, mem_size, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid =(iEnd + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    convert<<<blocksPerGrid, threadsPerBlock>>>(cuda_data_pointers.input_data, cuda_data_pointers.output_data, iEnd);
    
    CUDA_CALL(hipMemcpy(cpu_data_pointers.result_data, cuda_data_pointers.output_data, sizeof(my_float) * iEnd, hipMemcpyDeviceToHost));

    std::vector<my_float> aa(iEnd);
    for(int i = 0; i < iEnd; i++) {
        aa[i] = cpu_data_pointers.pointer_data_pwa_paras_type[i].wu[0] + cpu_data_pointers.pointer_data_pwa_paras_type[i].wu[1] + cpu_data_pointers.pointer_data_pwa_paras_type[i].wu[2] + cpu_data_pointers.pointer_data_pwa_paras_type[i].wu[3];
    }
    for(int i = 0; i < iEnd; i++)
    {
        if(cpu_data_pointers.result_data[i]-aa[i] != 0.0) {std::cout << "test failed!!!!!!! the result is not same!!"<< std::endl; return 0;}
    }
    std::cout << "test finish !!!! the result is same!" << std::endl;
    return 0;
}

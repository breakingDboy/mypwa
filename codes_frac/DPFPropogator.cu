#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"
#include "DPFPropogator.h"
#include "calEva.h"
const double rk=0.493677;
const double rp=0.139556995;

__device__ TComplex cro(my_float sx, my_float am1, my_float am2)
  {
	           TComplex ci = make_complex(0.0,1.0);
	           my_float t1=pow((am1+am2),2);
	           my_float t2=pow((am1-am2),2);
	           my_float st=(sx-t1)*(sx-t2);
	           my_float cro=sqrt(fabs(st))/sx;
		   TComplex result = make_complex(cro,0.0);
	           if (st<0.) result=cuCmuldc(cro,ci);
	           return  result;
  }
__device__ TComplex propogator980(my_float mass, my_float g11, my_float g22,my_float sx)
  {
	           TComplex ci = make_complex(0.0,1.0);
	           my_float rm=mass*mass;
               //TComplex propogator980=1.0/(rm-sx-ci*(g11*cro(sx,rp,rp)+g22*cro(sx,rk,rk)));
	           TComplex propogator980=cuCdivdc(1.0,(cuCsubdc((rm-sx),hipCmul(ci,hipCadd( cuCmuldc(g11,cro(sx,rp,rp)),cuCmuldc(g22,cro(sx,rk,rk)) )) )) );
	           return propogator980;
  }
__device__ TComplex pip(my_float sx)
  {
	           TComplex ci = make_complex(0.0,1.0);
	           my_float xk2=sx-0.3116676;     //0.3116676=16.*0.139568*0.139568
		   if(xk2<=0.)xk2=0.0;
	           my_float r4pip=sqrt(xk2/sx)/(1.0+exp(9.8-3.5*sx));    //9.8=3.5*2.8
	           return  make_complex(r4pip,0.0);
  }
__device__ TComplex propogator600(my_float mass, my_float b1, my_float b2, my_float b3, my_float b4, my_float b5, my_float sx)
  {
	           TComplex ci = make_complex(0.0,1.0);
      my_float am1=mass;
	           my_float as=am1*am1;
	           //my_float cgam1=(am1*(b1+b2*sx)*cro(sx,rp,rp)/cro(as,rp,rp)*(my_float)(sx-0.0097)/(my_float)(as-0.0097)*(my_float)exp(-(sx-as)/b3)).real();
              my_float cgam1=hipCreal(cuCmulcd(hipCdiv(cuCmuldc(am1*(b1+b2*sx),cro(sx,rp,rp)) , cro(as,rp,rp)) ,((my_float)(sx-0.0097)/(my_float)(as-0.0097)*(my_float)exp(-(sx-as)/b3))) );
	           my_float cgam2=hipCreal(hipCdiv( cuCmuldc((am1*b4),pip(sx)) , pip(as) ));
	           //TComplex propogator600=make_complex(1.0,0.0)/(as-sx-ci*b5*(cgam1+cgam2));
               TComplex propogator600=cuCdivdc(1.0,cuCsubdc((as-sx),cuCmulcd(ci,(b5*(cgam1+cgam2))) ) );
	           return propogator600;
	    }

__device__ TComplex propogator(my_float mass, my_float width, my_float sx)
{
	TComplex ci = make_complex(0.0,1.0);
	my_float am=mass;
	my_float g1=mass*width;
    //TComplex prop=g1/(sx-pow(am,2)+ci*g1);
	TComplex prop=cuCdivdc(g1,cuCadddc((sx-pow(am,2)),cuCmulcd(ci,g1)) );
	return prop;
}
__device__ TComplex propogator1270(my_float mass, my_float width, my_float sx)
{
	TComplex ci = make_complex(0.0,1.0);
	my_float rm=mass*mass;
	my_float gr=mass*width;
	my_float q2r=0.25*rm-0.0194792;
	my_float b2r=q2r*(q2r+0.1825)+0.033306;
	my_float g11270=gr*b2r/pow(q2r,2.5);
	my_float q2=0.25*sx-0.0194792;
	my_float b2=q2*(q2+0.1825)+0.033306;
	my_float g1=g11270*pow(q2,2.5)/b2;
	//TComplex prop=gr/(sx-rm+ci*g1);
    TComplex prop=cuCdivdc(gr,cuCadddc( (sx-rm),cuCmulcd(ci,g1)) );
	return prop;
}

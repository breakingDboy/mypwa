#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"
#include "cu_DPFPropogator.h"

const float rk=0.493677;
const float rp=0.139556995;

__device__ float2 cro(float sx, float am1, float am2)
  {
	           float2 ci = make_hipFloatComplex(0.0,1.0);
	           float t1=pow((am1+am2),2);
	           float t2=pow((am1-am2),2);
	           float st=(sx-t1)*(sx-t2);
	           float cro=sqrt(fabs(st))/sx;
		   float2 result = make_hipFloatComplex(cro,0.0);
	           if (st<0.) result=cuCmulfc(cro,ci);
	           return  result;
  }
__device__ float2 propogator980(float mass, float g11, float g22,float sx)
  {
	           float2 ci = make_hipFloatComplex(0.0,1.0);
	           float rm=mass*mass;
               //float2 propogator980=1.0/(rm-sx-ci*(g11*cro(sx,rp,rp)+g22*cro(sx,rk,rk)));
	           float2 propogator980=cuCdivfc(1.0,(cuCsubfc((rm-sx),hipCmulf(ci,hipCaddf( cuCmulfc(g11,cro(sx,rp,rp)),cuCmulfc(g22,cro(sx,rk,rk)) )) )) );
	           return propogator980;
  }
__device__ float2 pip(float sx)
  {
	           float2 ci = make_hipFloatComplex(0.0,1.0);
	           float xk2=sx-0.3116676;     //0.3116676=16.*0.139568*0.139568
		   if(xk2<=0.)xk2=0.0;
	           float r4pip=sqrt(xk2/sx)/(1.0+exp(9.8-3.5*sx));    //9.8=3.5*2.8
	           return  make_hipFloatComplex(r4pip,0.0);
  }
__device__ float2 propogator600(float mass, float b1, float b2, float b3, float b4, float b5, float sx)
  {
	           float2 ci = make_hipFloatComplex(0.0,1.0);
      float am1=mass;
	           float as=am1*am1;
	           //float cgam1=(am1*(b1+b2*sx)*cro(sx,rp,rp)/cro(as,rp,rp)*(float)(sx-0.0097)/(float)(as-0.0097)*(float)exp(-(sx-as)/b3)).real();
              float cgam1=hipCrealf(cuCmulcf(hipCdivf(cuCmulfc(am1*(b1+b2*sx),cro(sx,rp,rp)) , cro(as,rp,rp)) ,((float)(sx-0.0097)/(float)(as-0.0097)*(float)exp(-(sx-as)/b3))) );
	           float cgam2=hipCrealf(hipCdivf( cuCmulfc((am1*b4),pip(sx)) , pip(as) ));
	           //float2 propogator600=make_hipFloatComplex(1.0,0.0)/(as-sx-ci*b5*(cgam1+cgam2));
               float2 propogator600=cuCdivfc(1.0,cuCsubfc((as-sx),cuCmulcf(ci,(b5*(cgam1+cgam2))) ) );
	           return propogator600;
	    }

__device__ float2 propogator(float mass, float width, float sx)
{
	float2 ci = make_hipFloatComplex(0.0,1.0);
	float am=mass;
	float g1=mass*width;
    //float2 prop=g1/(sx-pow(am,2)+ci*g1);
	float2 prop=cuCdivfc(g1,cuCaddfc((sx-pow(am,2)),cuCmulcf(ci,g1)) );
	return prop;
}
__device__ float2 propogator1270(float mass, float width, float sx)
{
	float2 ci = make_hipFloatComplex(0.0,1.0);
	float rm=mass*mass;
	float gr=mass*width;
	float q2r=0.25*rm-0.0194792;
	float b2r=q2r*(q2r+0.1825)+0.033306;
	float g11270=gr*b2r/pow(q2r,2.5f);
	float q2=0.25*sx-0.0194792;
	float b2=q2*(q2+0.1825)+0.033306;
	float g1=g11270*pow(q2,2.5f)/b2;
	//float2 prop=gr/(sx-rm+ci*g1);
    float2 prop=cuCdivfc(gr,cuCaddfc( (sx-rm),cuCmulcf(ci,g1)) );
	return prop;
}

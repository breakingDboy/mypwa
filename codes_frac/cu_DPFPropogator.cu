#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"
#include "cu_DPFPropogator.h"

const double rk=0.493677;
const double rp=0.139556995;

__device__ double2 cro(double sx, double am1, double am2)
  {
	           double2 ci = make_hipDoubleComplex(0.0,1.0);
	           double t1=pow((am1+am2),2);
	           double t2=pow((am1-am2),2);
	           double st=(sx-t1)*(sx-t2);
	           double cro=sqrt(fabs(st))/sx;
		   double2 result = make_hipDoubleComplex(cro,0.0);
	           if (st<0.) result=cuCmuldc(cro,ci);
	           return  result;
  }
__device__ double2 propogator980(double mass, double g11, double g22,double sx)
  {
	           double2 ci = make_hipDoubleComplex(0.0,1.0);
	           double rm=mass*mass;
               //double2 propogator980=1.0/(rm-sx-ci*(g11*cro(sx,rp,rp)+g22*cro(sx,rk,rk)));
	           double2 propogator980=cuCdivdc(1.0,(cuCsubdc((rm-sx),hipCmul(ci,hipCadd( cuCmuldc(g11,cro(sx,rp,rp)),cuCmuldc(g22,cro(sx,rk,rk)) )) )) );
	           return propogator980;
  }
__device__ double2 pip(double sx)
  {
	           double2 ci = make_hipDoubleComplex(0.0,1.0);
	           double xk2=sx-0.3116676;     //0.3116676=16.*0.139568*0.139568
		   if(xk2<=0.)xk2=0.0;
	           double r4pip=sqrt(xk2/sx)/(1.0+exp(9.8-3.5*sx));    //9.8=3.5*2.8
	           return  make_hipDoubleComplex(r4pip,0.0);
  }
__device__ double2 propogator600(double mass, double b1, double b2, double b3, double b4, double b5, double sx)
  {
	           double2 ci = make_hipDoubleComplex(0.0,1.0);
      double am1=mass;
	           double as=am1*am1;
	           //double cgam1=(am1*(b1+b2*sx)*cro(sx,rp,rp)/cro(as,rp,rp)*(double)(sx-0.0097)/(double)(as-0.0097)*(double)exp(-(sx-as)/b3)).real();
              double cgam1=hipCreal(cuCmulcd(hipCdiv(cuCmuldc(am1*(b1+b2*sx),cro(sx,rp,rp)) , cro(as,rp,rp)) ,((double)(sx-0.0097)/(double)(as-0.0097)*(double)exp(-(sx-as)/b3))) );
	           double cgam2=hipCreal(hipCdiv( cuCmuldc((am1*b4),pip(sx)) , pip(as) ));
	           //double2 propogator600=make_hipDoubleComplex(1.0,0.0)/(as-sx-ci*b5*(cgam1+cgam2));
               double2 propogator600=cuCdivdc(1.0,cuCsubdc((as-sx),cuCmulcd(ci,(b5*(cgam1+cgam2))) ) );
	           return propogator600;
	    }

__device__ double2 propogator(double mass, double width, double sx)
{
	double2 ci = make_hipDoubleComplex(0.0,1.0);
	double am=mass;
	double g1=mass*width;
    //double2 prop=g1/(sx-pow(am,2)+ci*g1);
	double2 prop=cuCdivdc(g1,cuCadddc((sx-pow(am,2)),cuCmulcd(ci,g1)) );
	return prop;
}
__device__ double2 propogator1270(double mass, double width, double sx)
{
	double2 ci = make_hipDoubleComplex(0.0,1.0);
	double rm=mass*mass;
	double gr=mass*width;
	double q2r=0.25*rm-0.0194792;
	double b2r=q2r*(q2r+0.1825)+0.033306;
	double g11270=gr*b2r/pow(q2r,2.5);
	double q2=0.25*sx-0.0194792;
	double b2=q2*(q2+0.1825)+0.033306;
	double g1=g11270*pow(q2,2.5)/b2;
	//double2 prop=gr/(sx-rm+ci*g1);
    double2 prop=cuCdivdc(gr,cuCadddc( (sx-rm),cuCmulcd(ci,g1)) );
	return prop;
}
